#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <vector>
#include "hip/hip_vector_types.h"
#include <unistd.h>
#include <time.h>
#include "hip/device_functions.h"
#include "hip/hip_runtime.h"


#define fix1 3.14 / (60 * 180)

using namespace std;

__global__ void angles(volatile float *a0, volatile float *b0, volatile float *a1, volatile float*b1, volatile int *hist, volatile int* hist_r, volatile int* hist_s)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float ac, bb0, sb1, sb0, cb0, k, bb1, ssb1, cb1, ccb1, pom, minus, factorial, factorials, fb, fsb, k1, pom1, ccd;
	int angle;
	float fix2 = 57;

	bb0 = b0[idx];
	bb1 = b1[idx];
	ssb1 = sin(bb1);
	sb0 = sin(bb0);
	cb0 = cos(bb0);
	cb1 = cos(bb1);

	__shared__ int mn[720], r[720], s[720];

	if (threadIdx.x == 0) {
		for (int i = 0; i < 720; i++) {
			mn[i] = 0;
			r[i] = 0;
			s[i] = 0;
		}
	}

	__syncthreads();

	if (idx < 100000) {
		for (int i = 0; i < 100000; i++) {
			k = b1[i];
			k1 = a0[i] - a0[idx];
			pom = k * k;
			pom1 = k1 * k1;
			minus = -1;
			sb1 = k;
			ccb1 = 1;
			ccd = 1;
			fb = 3;
			fsb = 4;
			factorial = 2;
			factorials = 6;

			for (int i = 0; i < 5; i++) { 
				ccb1 = ccb1 + minus * pom / factorial;
				sb1 = sb1 + minus * k * pom / factorials;
				ccd = ccd + minus * pom1 / factorial;
				factorial = factorial * fb * (fb + 1);
				fb += 2;
				factorials = factorials * fsb * (fsb + 1);
				fsb += 2;
				minus = minus * (-1);
				pom = pom * k * k;
			}

			ac = acosf(sb0 * sb1 + cb0 * ccb1 * ccd);
			ac = (ac * fix2 / 0.25);

			angle = (int)ac;
			atomicAdd(&mn[angle], 1);
		}

		for (int i = idx + 1; i < 100000; i++) {
			k = b0[i];
			k1 = a0[i] - a0[idx];
			pom = k * k;
			pom1 = k1 * k1;
			minus = -1;
			sb1 = k;
			ccb1 = 1;
			ccd = 1;
			fb = 3;
			fsb = 4;
			factorial = 2;
			factorials = 6;

			for (int i = 0; i < 5; i++) {
				ccb1 = ccb1 + minus * pom / factorial;
				sb1 = sb1 + minus * k * pom / factorials;
				ccd = ccd + minus * pom1 / factorial;
				factorial = factorial * fb * (fb + 1);
				fb += 2;
				factorials = factorials * fsb * (fsb + 1);
				fsb += 2;
				minus = minus * (-1);
				pom = pom * k * k;
			}

			ac = acosf(sb0 * sb1 + cb0 * ccb1 * ccd);
			ac = (ac * fix2 / 0.25);
			angle = (int)ac;
			atomicAdd(&r[angle], 2);

			k = b1[i];
			k1 = a0[i] - a0[idx];
			pom = k * k;
			pom1 = k1 * k1;
			minus = -1;
			sb1 = k;
			ccb1 = 1;
			ccd = 1;
			fb = 3;
			fsb = 4;
			factorial = 2;
			factorials = 6;

			for (int i = 0; i < 5; i++) {
				ccb1 = ccb1 + minus * pom / factorial;
				sb1 = sb1 + minus * k * pom / factorials;
				ccd = ccd + minus * pom1 / factorial;
				factorial = factorial * fb * (fb + 1);
				fb += 2;
				factorials = factorials * fsb * (fsb + 1);
				fsb += 2;
				minus = minus * (-1);
				pom = pom * k * k;
			}

			ac = acosf((ssb1 * sb1) + cb1 * ccb1 * ccd);
			ac = (ac * fix2 / 0.25);
			angle = (int)ac;
			atomicAdd(&s[angle], 2);
		}
	}

	__syncthreads();

	if (threadIdx.x == 0) {
		for (int i = 0; i < 720; i++) {  
			hist[i + (blockIdx.x * 720)] = mn[i];
			hist_r[i + (blockIdx.x * 720)] = r[i];
			hist_s[i + (blockIdx.x * 720)] = s[i];
		}
	}
}

int main(int argc, char *argv[]) {
	FILE *real_g;
	FILE *synthetic_g;
	int galaxies_r, galaxies_s;
	clock_t start, end;

	start = clock();
	real_g = fopen("data_100k_arcmin.txt", "r");
	synthetic_g = fopen("flat_100k_arcmin.txt", "r");
	fscanf(real_g, "%d", &galaxies_r);
	fscanf(synthetic_g, "%d", &galaxies_s);

	int N = 100000;
	int xx = 782;
	size_t arraybytes = N * sizeof(float);
	size_t arraybytes1 = xx * 720 * sizeof(int);
	size_t arraybytes11 = xx * 720 * sizeof(float);
	size_t l = 720 * sizeof(int);
	size_t l1 = 720  * sizeof(float); // Check this

	// Allocate input vectors h_A and h_B in host memory
	float* h_A = (float*)malloc(arraybytes);
	float* h_B = (float*)malloc(arraybytes);
	float* h_A1 = (float*)malloc(arraybytes);
	float* h_B1 = (float*)malloc(arraybytes);
	int* h_C = (int*)malloc(arraybytes1);
	int* h_D = (int*)malloc(arraybytes1);
	int* h_E = (int*)malloc(arraybytes1);

	int* result = (int*)malloc(l);
	int* result_r = (int*)malloc(l);
	int* result_s = (int*)malloc(l);
	float* final = (float*)malloc(l1);

	for(int i = 0; i < galaxies_r; i++) {
		fscanf(real_g, "%e %e", &h_A[i], &h_B[i]);
		fscanf(synthetic_g, "%e %e", &h_A1[i], &h_B1[i]);

		h_A[i] = h_A[i] * fix1;
		h_A1[i] = h_A1[i] * fix1;
		h_B[i] = h_B[i] * fix1;
		h_B1[i] = h_B1[i] * fix1;
	}

	fclose(real_g);
	fclose(synthetic_g);

	float* d_A; hipMalloc(&d_A, arraybytes);
	float* d_B; hipMalloc(&d_B, arraybytes);
	float* d_A1; hipMalloc(&d_A1, arraybytes);
	float* d_B1; hipMalloc(&d_B1, arraybytes);
	int* d_C; hipMalloc(&d_C, arraybytes1);
	int* d_D; hipMalloc(&d_D, arraybytes1);
	int* d_E; hipMalloc(&d_E, arraybytes1);
	int* d_result; hipMalloc(&d_result, l);
	int* d_result_r; hipMalloc(&d_result_r, l);
	int* d_result_s; hipMalloc(&d_result_s, l);

	// Copy arrays from host memory to device memory
	hipMemcpy(d_A, h_A, arraybytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, arraybytes, hipMemcpyHostToDevice);
	hipMemcpy(d_A1, h_A1, arraybytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B1, h_B1, arraybytes, hipMemcpyHostToDevice);

	// Invoke kernel
	dim3 threadsPerBlock(128);
	dim3 threadsPerBlock1(736);
	dim3 blocksize2(1);
	dim3 blocksPerGrid(xx);

	double cpu_time_used;

	hipMemset(d_C, 0, arraybytes1);
	hipMemset(d_D, 0, arraybytes1);
	hipMemset(d_E, 0, arraybytes1);

	angles<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_A1, d_B1, d_C, d_D, d_E);

	hipMemcpy(h_C, d_C, arraybytes1, hipMemcpyDeviceToHost);
	hipMemcpy(h_D, d_D, arraybytes1, hipMemcpyDeviceToHost);
	hipMemcpy(h_E, d_E, arraybytes1, hipMemcpyDeviceToHost);

	for (int i = 0; i < 720 * xx; i++) {
		result[i % 720] += h_C[i];
		result_r[i % 720] += h_D[i];
		result_s[i % 720] += h_E[i];
	}

	result_r[0] = result_r[0] + 100000;
	result_s[0] = result_s[0] + 100000;

	final[0] = (float) ((float)(result_r[0] - 2 * result[0] + result_s[0] + 200000) / (float)(100000 + result_s[0]));
	for(int i = 1; i < 720; i++) {
		final[i] = (float) ((float)(result_r[i] - 2 * result[i] + result_s[i]) / (float) result_s[i]);
	}

	end = clock();
	cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;

	double brk = 0;
	printf("%f\n", cpu_time_used);

	for(int i = 0; i < 720; i++) {
		printf( "%f ", final[i]);
	}

	hipFree(d_A); hipFree(d_B); hipFree(d_C);
	hipFree(h_A); hipFree(h_B); hipFree(h_C);
	hipFree(d_D); hipFree(h_D); hipFree(d_E); hipFree(h_E);
	hipFree(d_A1); hipFree(h_A1); hipFree(d_B1); hipFree(h_B1);
}
