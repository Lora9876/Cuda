#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<unistd.h>

#include<hip/hip_runtime.h>

using namespace std;

#define SUBMATRIX_SIZE 16384

////////////////////////////////////////////////////////////////////////
// Number of histogram bins has to be edited by hand, prior to
// copmilation.
////////////////////////////////////////////////////////////////////////

#define DEFAULT_NBINS 254 
//#define DEFAULT_NBINS 126 
//#define DEFAULT_NBINS 62 
//#define DEFAULT_NBINS 30 

#define CONV_FACTOR 57.2957795 // 180/pi


int doCalcRaDec(FILE *infile0, FILE *infile1, FILE *outfile, bool silent_on_GPU_testing, float scale_factor, int nbins, float hist_lower_range, float hist_upper_range, float hist_bin_width, int log_binning_flag, bool two_different_files, float conv_factor_angle);

int doCalcMpc(FILE *infile0, FILE *infile1, FILE *outfile, bool silent_on_GPU_testing, float scale_factor, int nbins, float hist_lower_range, float hist_upper_range, float hist_bin_width, int log_binning_flag, bool two_different_files, float conv_factor_angle);

void getDeviceDiagnostics(int tot_Gals, int n_coords);

////////////////////////////////////////////////////////////////////////
// Kernel to calculate angular distances between galaxies and histogram
// the distances.
////////////////////////////////////////////////////////////////////////
__global__ void distance(volatile float *a0, volatile float *d0, volatile float *a1, volatile float *d1, int xind, int yind, int max_xind, int max_yind, volatile int *dev_hist, float hist_min, float hist_max, int nbins, float bin_width, int log_binning=0, bool two_different_files=1, float conv_factor_angle=57.2957795)
{

    ////////////////////////////////////////////////////////////////////////////
    // Idx will keep track of which thread is being calculated within a given 
    // warp.
    ////////////////////////////////////////////////////////////////////////////
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // This should range to SUBMATRIX_SIZE

    idx += xind;

    ////////////////////////////////////////////////////////////////////////
    // Shared memory stuff.
    ////////////////////////////////////////////////////////////////////////
    __shared__ int shared_hist[DEFAULT_NBINS+2];
    // Note that we only clear things out for the first thread on each block.
    if(threadIdx.x==0)
    {
        for (int i=0;i<nbins+2;i++)
            shared_hist[i] = 0;
    }
    __syncthreads();
    ////////////////////////////////////////////////////////////////////////

    if (idx<max_xind)
    {
        int i=0;

        float alpha_rad = a0[idx];
        float delta0 = d0[idx];
        float cos_d0 = cos(delta0);
        float sin_d0 = sin(delta0);
        float dist;

        int bin_index = 0; 

        float a_diff, sin_a_diff, cos_a_diff;
        float cos_d1, sin_d1, numer, denom, mult1, mult2;    
        float d1_rad;

        bool do_calc = 1;

        int ymax = yind + SUBMATRIX_SIZE;

        if (ymax>max_yind)
        {
            ymax = max_yind;
        }

        for(i=yind; i<ymax; i++)
        {
            if (two_different_files)
            {
                do_calc = 1;
            }
            else // Doing the same file
            {
                if(idx > i)
                    do_calc=1;
                else
                    do_calc=0;
            }
            //if(idx > i) ///////// CHECK THIS
            if (do_calc)
            {
                a_diff = a1[i] - alpha_rad;
                d1_rad = d1[i];

                sin_a_diff = sin(a_diff);
                cos_a_diff = cos(a_diff);

                sin_d1 = sin(d1_rad);
                cos_d1 = cos(d1_rad);

                mult1 = cos_d1 * cos_d1 * sin_a_diff * sin_a_diff;
                mult2 = cos_d0 * sin_d1 - sin_d0 * cos_d1 * cos_a_diff;
                mult2 = mult2 * mult2;

                numer = sqrt(mult1 + mult2); 

                denom = sin_d0 *sin_d1 + cos_d0 * cos_d1 * cos_a_diff;

                dist = atan2(numer,denom);  
                dist *= conv_factor_angle;  // Convert to degrees or what have you.

                if(dist < hist_min)
                    bin_index = 0; 
                else if(dist >= hist_max)
                    bin_index = nbins + 1;
                else
                {
                    if (log_binning==0)
                    {
                        bin_index = int((dist-hist_min)/bin_width) + 1;
                    }
                    else if (log_binning==1)// log binning
                    {
                        bin_index = int((log(dist)-log(hist_min))/bin_width) + 1;
                    }
                    else if (log_binning==2)// log 10 binning
                    {
                        bin_index = int((log10(dist)-log10(hist_min))/bin_width) + 1;
                    }
                }

                atomicAdd(&shared_hist[bin_index],1);

            }
        }
    }

    __syncthreads();

    if(threadIdx.x==0)
    {
        for(int i=0;i<nbins+2;i++)
            dev_hist[i+(blockIdx.x*(nbins+2))]=shared_hist[i];
    }

}







////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////
// Main
////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    // Needed for parsing command-line arguments.
    extern char *optarg;
    extern int optind, optopt, opterr;
    int c;
    char *infilename0 = NULL;
    char *infilename1 = NULL;
    char *outfilename = NULL;
    char defaultoutfilename[256];
    sprintf(defaultoutfilename,"default_out.dat");

    float hist_lower_range = 0.0000001;
    float hist_upper_range = 0;
    int nbins = DEFAULT_NBINS;
    float hist_bin_width = 0.05;
    int log_binning_flag = 0; // False

    float scale_factor = 1.0; // For if we need to convert input to arcsec or arcmin
    float conv_factor_angle = 57.2957795; // 180/pi // For if we need to convert arcdistance to arcsec or arcmin
    int radec_input = 1; // are we using ra/dec coords, or x/y/z coords? 
    bool silent_on_GPU_testing = false;
    int cuda_device = 0;

    ////////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////////

    while ((c = getopt(argc, argv, "ao:L:l:w:smSd:pD")) != -1) {
        switch(c) {
            case 'L':
                printf("L is set\n");
                hist_lower_range = atof(optarg);
                break;
            case 'w':
                hist_bin_width = atof(optarg);
                printf("Histogram bin width: %f\n",hist_bin_width);
                break;
            case 'l':
                log_binning_flag = atoi(optarg);
                printf("Will use log binning.\n");
                break;
            case 'D':
                scale_factor = 1./(2*3.14159/360.); // To convert degrees to radians.
                conv_factor_angle *= 1.0; // Convert radians to degrees.
                printf("Reading in values assuming they are degrees.\n");
                printf("scale_factor: %f\n",scale_factor);
                printf("conv_factor_angle: %f\n",conv_factor_angle);
                break;
            case 's':
                scale_factor = 206264.0; // To convert arcseconds to radians.
                conv_factor_angle *= 3600.0; // convert radians to arcseconds.
                printf("Reading in values assuming they are arcseconds.\n");
                printf("scale_factor: %f\n",scale_factor);
                printf("conv_factor_angle: %f\n",conv_factor_angle);
                break;
            case 'm':
                scale_factor = 3437.74677; // To convert arcminutes to radians.
                conv_factor_angle *= 60.0; // convert radians to arcminutes.
                printf("scale_factor: %f\n",scale_factor);
                printf("conv_factor_angle: %f\n",conv_factor_angle);
                printf("Reading in values assuming they are arcminutes.\n");
                break;
            case 'o':
                outfilename = optarg;
                printf("Output filename is %s\n", outfilename);
                break;
            case 'd':
                cuda_device = atoi(optarg); // Use this CUDA device.
                printf("Will attempt to use CUDA device %d\n",cuda_device);
                break;
            case 'S':
                printf("Silent mode - don't run the GPU test (suppresses some output)\n");
                silent_on_GPU_testing = true;
                break;
            case 'p':
                printf("Using input files in Mpc format");
                radec_input = 0;
                break;
            case '?':
                printf("unknown arg %c\n", optopt);
                break;
        }
    }

    if (argc < 2)
    {

        printf("\nMust pass in at least two input files on command line!\n");
        printf("\nUsage: ", argv[0] );
        //printf(" <cluster_data file> <distances file> \n\n");
        exit(1);
    }

    // Set a default output file name, if none was passed in on the 
    // command line.
    if (outfilename == NULL) 
    {
        outfilename = defaultoutfilename;
        printf("Output filename is %s\n", outfilename);
    }

    float temp_lo = hist_lower_range;
    if (hist_upper_range == 0)
    {
        if (log_binning_flag==0)
        {
            for (int i=0;i<nbins;i++)
            {
                hist_upper_range = temp_lo + hist_bin_width;
                temp_lo = hist_upper_range;
            }
        }
        else if (log_binning_flag==1)
        {
            for (int i=0;i<nbins;i++)
            {
                hist_upper_range = exp(log(temp_lo) + hist_bin_width);
                temp_lo = hist_upper_range;
            }
        }
        else if (log_binning_flag==2)
        {
            for (int i=0;i<nbins;i++)
            {
                hist_upper_range = pow(10,(log10(temp_lo) + hist_bin_width));
                temp_lo = hist_upper_range;
            }
        }
    }
    printf("hist_upper_range: %f\n",hist_upper_range);

    FILE *infile0, *infile1, *outfile ;

    infile0 = fopen(argv[optind],"r");
    infile1 = fopen(argv[optind+1],"r");

    printf("Opening input file 0: %s\n",argv[optind]);
    printf("Opening input file 1: %s\n",argv[optind+1]);

    outfile = fopen(outfilename, "w");

    ////////////////////////////////////////////////////////////////////////////
    // Check to see if the two files are actually the same file.
    // This is the case for the DD and RR calculations and change slightly
    // the exact calculations being performed.
    ////////////////////////////////////////////////////////////////////////////
    bool two_different_files = 1;
    if (strcmp(argv[optind],argv[optind+1])==0)
    {
        two_different_files = 0;
        printf("Using the same file!\n");
    }
    printf("\n");

    ////////////////////////////////////////////////////////////////////////
    // Set the CUDA device. This is useful if your machine has multiple GPUs
    // on it. 
    ////////////////////////////////////////////////////////////////////////
    hipError_t error_id = hipSetDevice(cuda_device);
    if (error_id == hipSuccess) {
        printf( "hipSetDevice returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id) );
    }
    else{
        printf( "hipSetDevice failed on Device %d!\n\n",cuda_device);
        exit(-1);
    }

    if(radec_input==1) int success = doCalcRaDec(infile0, infile1, outfile, silent_on_GPU_testing, scale_factor, nbins, hist_lower_range, hist_upper_range, hist_bin_width, log_binning_flag, two_different_files, conv_factor_angle);

    else  int success = doCalcMpc(infile0, infile1, outfile, silent_on_GPU_testing, scale_factor, nbins, hist_lower_range, hist_upper_range, hist_bin_width, log_binning_flag, two_different_files, conv_factor_angle);

}






////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////
// Module that runs the calculations for input files in the format ra:dec
////////////////////////////////////////////////////////////////////////

int doCalcRaDec(FILE *infile0, FILE *infile1, FILE *outfile, bool silent_on_GPU_testing, float scale_factor, int nbins, float hist_lower_range, float hist_upper_range, float hist_bin_width, int log_binning_flag, bool two_different_files, float conv_factor_angle){
    
    float *d_alpha0, *d_delta0;
    float *h_alpha0, *h_delta0;

    float *d_alpha1, *d_delta1;
    float *h_alpha1, *h_delta1;

    int NUM_GALAXIES0;
    int NUM_GALAXIES1;

    //////////////////////////////////////////////////////////////////////
    // Read in the galaxy files.
    ////////////////////////////////////////////////////////////////////////////
    // Read in the first file
    ////////////////////////////////////////////////////////////////////////////

    fscanf(infile0, "%d", &NUM_GALAXIES0);

    int size_of_galaxy_array0 = NUM_GALAXIES0 * sizeof(float);    

    printf("SIZE 0 # GALAXIES: %d\n",NUM_GALAXIES0);

    h_alpha0 = (float*)malloc(size_of_galaxy_array0);
    h_delta0 = (float*)malloc(size_of_galaxy_array0);
    float temp0, temp1, tempdum;

    for(int i=0; i<NUM_GALAXIES0; i++)
    {
        //fscanf(infile0, "%f %f", &temp0, &temp1);
        fscanf(infile0, "%e %e %e %e %e %e", &temp0, &temp1, &tempdum, &tempdum, &tempdum, &tempdum);
        h_alpha0[i] = temp0/scale_factor;
        h_delta0[i] = temp1/scale_factor;
        if (i<10)
        printf("%e %e\n", h_alpha0[i], h_delta0[i]);
    }

    //exit(1);

    ////////////////////////////////////////////////////////////////////////////
    // Read in the second file
    ////////////////////////////////////////////////////////////////////////////

    fscanf(infile1, "%d", &NUM_GALAXIES1);
    int size_of_galaxy_array1 = NUM_GALAXIES1 * sizeof(float);    
    printf("SIZE 1 # GALAXIES: %d\n",NUM_GALAXIES1);

    h_alpha1 = (float*)malloc(size_of_galaxy_array1);
    h_delta1 = (float*)malloc(size_of_galaxy_array1);

    for(int i=0; i<NUM_GALAXIES1; i++)
    {
        //fscanf(infile1, "%f %f", &temp0, &temp1);
        fscanf(infile1, "%e %e %e %e %e %e", &temp0, &temp1, &tempdum, &tempdum, &tempdum, &tempdum);
        h_alpha1[i] = temp0/scale_factor;
        h_delta1[i] = temp1/scale_factor;
        if (i<10)
        printf("%e %e\n", h_alpha1[i], h_delta1[i]);
    }

//get device diagnostics
 if (!silent_on_GPU_testing) getDeviceDiagnostics(NUM_GALAXIES0+NUM_GALAXIES1, 2);

    ////////////////////////////////////////////////////////////////////////////
    // Allocation of histogram
    ///////////////////////////////////////////////////////////////////////////

    int *hist, *dev_hist;

    int size_hist = SUBMATRIX_SIZE * (nbins+2);
    int size_hist_bytes = size_hist*sizeof(int);

    hist = (int*)malloc(size_hist_bytes);
    memset(hist, 0, size_hist_bytes);

    printf("Size of histogram: %d bytes\n",size_hist_bytes);
    hipMalloc((void **) &dev_hist, (size_hist_bytes));
    hipMemset(dev_hist, 0, size_hist_bytes);

    unsigned long  *hist_array;

    int hist_array_size = (nbins+2) * sizeof(unsigned long);
    hist_array =  (unsigned long*)malloc(hist_array_size);
    printf("Size of histogram array: %d bytes\n",hist_array_size);
    memset(hist_array,0,hist_array_size); 

    ////////////////////////////////////////////////////////////////////////////
    // Define the grid and block size
    ////////////////////////////////////////////////////////////////////////////
    dim3 grid, block;
    // 128*4 = 512, the amount of memory needed for one histogram.
    // 8192*4 = 32768 is max memory to ask for for the histograms.
    // 8192/128 = 64, is is the right number of blocks?
    grid.x = 8192/(DEFAULT_NBINS+2); // Is this the number of blocks?
    block.x = SUBMATRIX_SIZE/grid.x; // Is this the number of threads per block? NUM_GALAXIES/block.x;
    // SUBMATRIX is the number of threads per warp? Per kernel call?
    ////////////////////////////////////////////////////////////////////////////

    hipMalloc((void **) &d_alpha0, size_of_galaxy_array0 );
    hipMalloc((void **) &d_delta0, size_of_galaxy_array0 );

    hipMalloc((void **) &d_alpha1, size_of_galaxy_array1 );
    hipMalloc((void **) &d_delta1, size_of_galaxy_array1 );

    // Check to see if we allocated enough memory.
    if (0==d_alpha0 || 0==d_delta0 || 0==d_alpha1 || 0==d_delta1 || 0==dev_hist)
    {
        printf("couldn't allocate memory\n");
        return 1;
    }

    // Initialize array to all 0's
    hipMemset(d_alpha0,0,size_of_galaxy_array0);
    hipMemset(d_delta0,0,size_of_galaxy_array0);
    hipMemset(d_alpha1,0,size_of_galaxy_array1);
    hipMemset(d_delta1,0,size_of_galaxy_array1);

    hipMemcpy(d_alpha0, h_alpha0, size_of_galaxy_array0, hipMemcpyHostToDevice );
    hipMemcpy(d_delta0, h_delta0, size_of_galaxy_array0, hipMemcpyHostToDevice );
    hipMemcpy(d_alpha1, h_alpha1, size_of_galaxy_array1, hipMemcpyHostToDevice );
    hipMemcpy(d_delta1, h_delta1, size_of_galaxy_array1, hipMemcpyHostToDevice );

    int x, y;

    int num_submatrices_x = NUM_GALAXIES0 / SUBMATRIX_SIZE;
    int num_submatrices_y = NUM_GALAXIES1 / SUBMATRIX_SIZE;
    
    // Take care of edges of matrix.
    if (NUM_GALAXIES0%SUBMATRIX_SIZE != 0)
    {
        num_submatrices_x += 1;
    }
    if (NUM_GALAXIES1%SUBMATRIX_SIZE != 0)
    {
        num_submatrices_y += 1;
    }


    printf("Breaking down the calculations.\n");
    printf("Number of submatrices: %dx%d\n",num_submatrices_x,num_submatrices_y);
    printf("Number of calculations per submatrices: %dx%d\n",SUBMATRIX_SIZE,SUBMATRIX_SIZE);

    int bin_index = 0;
    for(int k = 0; k < num_submatrices_y; k++)
    {
        y = k*SUBMATRIX_SIZE;
        //printf("%d %d\n",k,y);
        for(int j = 0; j < num_submatrices_x; j++)
        {
            x = j*SUBMATRIX_SIZE; 

            //printf("----\n");
            //printf("%d %d\t\t%d %d\n",k,y,j,x);
            //printf("----\n");

            // Set the histogram to all zeros each time.
            hipMemset(dev_hist,0,size_hist_bytes);

            int max_x = NUM_GALAXIES0;
            int max_y = NUM_GALAXIES1;

            distance<<<grid,block>>>(d_alpha0, d_delta0,d_alpha1, d_delta1, x, y, max_x, max_y, dev_hist, hist_lower_range, hist_upper_range, nbins, hist_bin_width, log_binning_flag, two_different_files,conv_factor_angle);
            hipMemcpy(hist, dev_hist, size_hist_bytes, hipMemcpyDeviceToHost);

            ////////////////////////////////////////////////////////////////////
            // Sum up the histograms from each thread (hist).
            ////////////////////////////////////////////////////////////////////
            for(int m=0; m<size_hist; m++)
            {
                bin_index = m%(nbins+2);
                hist_array[bin_index] += hist[m];
            }    
        }  
    }

    unsigned long total = 0;

    float lo = hist_lower_range;
    float hi = 0;
    for(int k=0; k<nbins+1; k++)
    {
        if (k==0)
        {
            //fprintf(outfile, "Underflow below %.3e %s %lu \n", lo, ",",  hist_array[k]);
        }
        else
        {
            if (log_binning_flag==0)
            {
                hi = lo + hist_bin_width;
            }
            else if (log_binning_flag==1)
            {
                //printf("lo: %f\t\tlog(lo): %f\n",lo,log(lo));
                hi = exp(log(lo) + hist_bin_width);
            }
            else if (log_binning_flag==2)
            {
                //printf("lo: %f\t\tlog10(lo): %f\n",lo,log10(lo));
                hi = pow(10,(log10(lo) + hist_bin_width));
            }

            fprintf(outfile, "%.3e %.3e %lu \n",lo,hi,hist_array[k]);
            total += hist_array[k];

            lo = hi;
        }
    }
    printf("total: %lu \n", total);

    fclose(infile0);
    fclose(infile1);
    fclose(outfile);

    free(h_alpha0);
    free(h_delta0);
    free(h_alpha1);
    free(h_delta1);
    free(hist);

    hipFree(d_alpha0);
    hipFree(d_delta0);  
    hipFree(d_alpha1);
    hipFree(d_delta1);  
    hipFree(dev_hist);

    return 0;
}  
//////////////////////////////////////////////////////////////////////



