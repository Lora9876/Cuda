#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<unistd.h>
#include <time.h>
#include "hip/device_functions.h"
#include "hip/hip_runtime.h"

#include <time.h>

using namespace std;

__global__ void angles(volatile float *a0, volatile float *b0, volatile float *a1, volatile float*b1, volatile int *hist, volatile int* hist_r, volatile int* hist_s)

{
	int idxx = blockIdx.x * blockDim.x + threadIdx.x; 
	int idy =  threadIdx.y; 
	
	int idx;
	idx=idxx*1024 +idy; 

	float ac;//721? koliko puta ucitavas i gde  da mnozis...zasto float proveri koliko imas preracunavanja
    int angle; float fix1=3.14/(60*180); float fix2=57;
    
   
    __shared__ int mn[720], r[720], s[720];
    if((threadIdx.x==0) && (threadIdx.y==0))
    {
        for (int i=0;i<720;i++)
	{ mn[i] = 0; r[i]=0;s[i]=0;} 
    }
    __syncthreads();


    if (idx<10000)
    {
      
        for(int i=0; i<100000; i++)
        	{
            ac= acosf((sin(b0[idx]*fix1)*sin(b1[i]*fix1))+ cos(b0[idx]*fix1)*cos(b1[i]*fix1)*cos((a1[i]-a0[idx])*fix1));
		ac= (ac*fix2/0.25); 
		angle=(int) ac; 
             atomicAdd(&mn[angle],1);
		}
	  /*  for(int i=idx+1; i<100000;i++)
	    {  ac= acosf((sin(b0[idx]*fix1)*sin(b0[i]*fix1))+ cos(b0[idx]*fix1)*cos(b0[i]*fix1)*cos((a0[i]-a0[idx])*fix1));
	    ac= (ac*fix2/0.25); 
            angle=(int) ac; 
            atomicAdd(&r[angle],1);
	     
            ac= acosf((sin(b1[idx]*fix1)*sin(b1[i]*fix1))+ cos(b1[idx]*fix1)*cos(b1[i]*fix1)*cos((a1[idx]-a1[i])*fix1));
            ac= (ac*fix2/0.25); 
	    angle=(int) ac; 
            atomicAdd(&s[angle],1);

	
	    
               

                }*/
	
            }
        
    

    __syncthreads();

      if((threadIdx.x==0) && (threadIdx.y==0))
    {
        for(int i=0;i<720;i++)
	{ hist[i+(blockIdx.x*720)]=mn[i]; hist_r[i+(blockIdx.x*720)]=r[i]; hist_s[i+(blockIdx.x*720)]=s[i];}
    }

}

int main(int argc, char *argv[])
{
FILE *real_g; FILE *synthetic_g;
int galaxies_r, galaxies_s; 
clock_t start, end;
start = clock();
	real_g = fopen("data_100k_arcmin.txt","r");
    	synthetic_g = fopen("flat_100k_arcmin.txt","r");	
	fscanf(real_g, "%d", &galaxies_r);
	fscanf(synthetic_g,  "%d", &galaxies_s);
	

int N =100000;
size_t arraybytes = N * sizeof(float);
	size_t arraybytes1 =20 *720 *sizeof(int);
	size_t l=720*sizeof(int);
	size_t l1=720*sizeof(float);
// Allocate input vectors h_A and h_B in host memory
float* h_A = (float*)malloc(arraybytes);
float* h_B = (float*)malloc(arraybytes);
float* h_A1 = (float*)malloc(arraybytes);
float* h_B1 = (float*)malloc(arraybytes);
int* h_C = (int*)malloc(arraybytes1); 
int* h_D = (int*)malloc(arraybytes1); 	
int* h_E = (int*)malloc(arraybytes1); 	
	int* result=(int*)malloc(l); 
	int* result_r=(int*)malloc(l); 
	int* result_s=(int*)malloc(l); 
	float* final=(float*)malloc(l1); 
	for(int i=0; i<galaxies_r; i++)
    {
       
        fscanf(real_g, "%e %e", &h_A[i], &h_B[i]);
       fscanf(synthetic_g, "%e %e", &h_A1[i], &h_B1[i]);}
    fclose(real_g);
	 fclose(synthetic_g);	
	
float* d_A; hipMalloc(&d_A, arraybytes);
float* d_B; hipMalloc(&d_B, arraybytes);
float* d_A1; hipMalloc(&d_A1, arraybytes);
float* d_B1; hipMalloc(&d_B1, arraybytes);
int* d_C; hipMalloc(&d_C, arraybytes1);
	int* d_D; hipMalloc(&d_D, arraybytes1);
	int* d_E; hipMalloc(&d_E, arraybytes1);
// Copy arrays from host memory to device memory
hipMemcpy(d_A, h_A, arraybytes, hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B, arraybytes, hipMemcpyHostToDevice);
hipMemcpy(d_A1, h_A1, arraybytes, hipMemcpyHostToDevice);
hipMemcpy(d_B1, h_B1, arraybytes, hipMemcpyHostToDevice);
// Invoke kernel
	
    dim3 threadsPerBlock(1024,1024) ;
    int blocksPerGrid=20; 
     double cpu_time_used;
     
    
    hipMemset(d_C,0,arraybytes1);
	hipMemset(d_D,0,arraybytes1);
	hipMemset(d_E,0,arraybytes1);
		angles<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B,d_A1, d_B1, d_C,d_D,d_E);

      hipMemcpy(h_C, d_C, arraybytes1, hipMemcpyDeviceToHost);
	hipMemcpy(h_D, d_D, arraybytes1, hipMemcpyDeviceToHost);
	hipMemcpy(h_E, d_E, arraybytes1, hipMemcpyDeviceToHost);
	
	for(int i=0; i<720*20; i++)
	{	result[i%720]+= h_C[i];result_r[i%720]+=h_D[i];result_s[i%720]+=h_E[i];} 

		
	for(int i=0;i<720;i++)
		final[i]=(float) (result_r[i]-2*result[i]+result_s[i])/result_s[i]; 
	
	end = clock();
     cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
	
	int brk=0; 
	printf("%f\n", cpu_time_used); 
		for(int i=0; i<720; i++)
			brk+=result[i]; 
	printf("%d\n ", brk);
	brk=0;
	for(int i=0; i<720; i++)
			brk+=result_s[i]; 
	printf("%d\n ", brk);
	brk=0;
	for(int i=0; i<720; i++)
			brk+=result_r[i]; 
	printf("%d\n ", brk);
	brk=0;
		//{printf("%f ", final[i]);   }
	/*printf("\n druga\n " ) ; 
	for(int i=0; i<720; i++)
		{printf("%d ", result_r[i]);   }
	printf("\n treca\n " ) ; 
	for(int i=0; i<720; i++)
		{printf("%d ", result_s[i]);   }*/

hipFree(d_A); hipFree(d_B); hipFree(d_C);
	hipFree(h_A); hipFree(h_B); hipFree(h_C);hipFree(d_D);hipFree(h_D);hipFree(d_E);hipFree(h_E);
	hipFree(d_A1);hipFree(h_A1);hipFree(d_B1);hipFree(h_B1);

	
}








    
  
