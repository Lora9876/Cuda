#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include <vector>
#include "hip/hip_vector_types.h"
#include<unistd.h>
#include <time.h>
#include "hip/device_functions.h"
#include "hip/hip_runtime.h"

#include <time.h>



using namespace std;

__global__ void angles(volatile float *a0, volatile float *b0, volatile float *a1, volatile float*b1, volatile int *hist, volatile int* hist_r, volatile int* hist_s)

{
	int idx= blockIdx.x * blockDim.x + threadIdx.x; 
	
	float ac;//721? koliko puta ucitavas i gde  da mnozis...zasto float proveri koliko imas preracunavanja
    int angle; float fix1=3.14/(60*180); float fix2=57;
    
   
    __shared__ int mn[180], mn1[180], mn2[180], mn3[180], r[180], r1[180], r2[180], r3[180], s[180], s1[180], s2[180], s3[180];
   if(threadIdx.x==0 )
    {
        for (int i=0;i<720;i++)
	{ mn[i] = 0; r[i]=0;s[i]=0;  mn1[i] = 0; r1[i]=0;s1[i]=0; mn2[i] = 0; r2[i]=0;s2[i]=0; mn3[i] = 0; r3[i]=0;s3[i]=0;
	 } 
    }
    __syncthreads(); 
 

   if ( idx<100000)
    {
      
        for(int i=0; i<100000; i++)
        	{
		   
            ac= acosf((sin(b0[idx]*fix1)*sin(b1[i]*fix1))+ cos(b0[idx]*fix1)*cos(b1[i]*fix1)*cos((a1[i]-a0[idx])*fix1));
		ac= (ac*fix2/0.25); 
	
		angle=(int) ac; 
		if(ac>539) 
		  atomicAdd(&mn3[angle-540],1);
		else if(ac>359)
			atomicAdd(&mn2[angle-360],1);
		else if(ac>179)
			atomicAdd(&mn1[angle-179],1);
		else 
			atomicAdd(&mn[angle],1);
		}
		
	   for(int i=idx+1; i<100000;i++)
	    {  ac= acosf((sin(b0[idx]*fix1)*sin(b0[i]*fix1))+ cos(b0[idx]*fix1)*cos(b0[i]*fix1)*cos((a0[i]-a0[idx])*fix1));
	    ac= (ac*fix2/0.25); 
            angle=(int) ac; 
            if(ac>539) 
		  atomicAdd(&r3[angle-540],1);
		else if(ac>359)
			atomicAdd(&r2[angle-360],1);
		else if(ac>179)
			atomicAdd(&r1[angle-179],1);
		else 
			atomicAdd(&r[angle],1);
		
	     
            ac= acosf((sin(b1[idx]*fix1)*sin(b1[i]*fix1))+ cos(b1[idx]*fix1)*cos(b1[i]*fix1)*cos((a1[idx]-a1[i])*fix1));
            ac= (ac*fix2/0.25); 
	    angle=(int) ac; 
            if(ac>539) 
		  atomicAdd(&s3[angle-540],1);
		else if(ac>359)
			atomicAdd(&s2[angle-360],1);
		else if(ac>179)
			atomicAdd(&s1[angle-179],1);
		else 
			atomicAdd(&s[angle],1);
		

                }
	
           }
	

    __syncthreads();

      if(threadIdx.x==0)
    {
        for(int i=0;i<180;i++)
	{  
		hist[i+(blockIdx.x*720)]=mn[i]; hist_r[i+(blockIdx.x*720)]=r[i]; hist_s[i+(blockIdx.x*720)]=s[i];
		hist[i+180+(blockIdx.x*720)]=mn1[i]; hist_r[i+180+(blockIdx.x*720)]=r1[i]; hist_s[i+180+(blockIdx.x*720)]=s1[i];
		hist[i+360+(blockIdx.x*720)]=mn2[i]; hist_r[i+360+(blockIdx.x*720)]=r2[i]; hist_s[i+360+(blockIdx.x*720)]=s2[i];
		hist[i+540+(blockIdx.x*720)]=mn3[i]; hist_r[i+540+(blockIdx.x*720)]=r3[i]; hist_s[i+540+(blockIdx.x*720)]=s3[i];}
    }

}

int main(int argc, char *argv[])
{
FILE *real_g; FILE *synthetic_g;
int galaxies_r, galaxies_s; 
clock_t start, end;
start = clock();
	real_g = fopen("data_100k_arcmin.txt","r");
    	synthetic_g = fopen("flat_100k_arcmin.txt","r");	
	fscanf(real_g, "%d", &galaxies_r);
	fscanf(synthetic_g,  "%d", &galaxies_s);
	

int N =100000;
	int xx=530; 
size_t arraybytes = N * sizeof(float);
	size_t arraybytes1 =xx *720 *sizeof(int);
	size_t l=720*sizeof(int);
	size_t l1=720*sizeof(float);
// Allocate input vectors h_A and h_B in host memory
float* h_A = (float*)malloc(arraybytes);
float* h_B = (float*)malloc(arraybytes);
float* h_A1 = (float*)malloc(arraybytes);
float* h_B1 = (float*)malloc(arraybytes);
int* h_C = (int*)malloc(arraybytes1); 
int* h_D = (int*)malloc(arraybytes1); 	
int* h_E = (int*)malloc(arraybytes1); 	
	int* result=(int*)malloc(l); 
	int* result_r=(int*)malloc(l); 
	int* result_s=(int*)malloc(l); 
	float* final=(float*)malloc(l1); 
	for(int i=0; i<galaxies_r; i++)
    {
       
        fscanf(real_g, "%e %e", &h_A[i], &h_B[i]);
       fscanf(synthetic_g, "%e %e", &h_A1[i], &h_B1[i]);}
    fclose(real_g);
	 fclose(synthetic_g);	
	
float* d_A; hipMalloc(&d_A, arraybytes);
float* d_B; hipMalloc(&d_B, arraybytes);
float* d_A1; hipMalloc(&d_A1, arraybytes);
float* d_B1; hipMalloc(&d_B1, arraybytes);
int* d_C; hipMalloc(&d_C, arraybytes1);
	int* d_D; hipMalloc(&d_D, arraybytes1);
	int* d_E; hipMalloc(&d_E, arraybytes1);
// Copy arrays from host memory to device memory
hipMemcpy(d_A, h_A, arraybytes, hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B, arraybytes, hipMemcpyHostToDevice);
hipMemcpy(d_A1, h_A1, arraybytes, hipMemcpyHostToDevice);
hipMemcpy(d_B1, h_B1, arraybytes, hipMemcpyHostToDevice);
// Invoke kernel
	dim3 threadsPerBlock(192);
	
 
    dim3 blocksPerGrid(xx); 
     double cpu_time_used;
     
    
    hipMemset(d_C,0,arraybytes1);
	hipMemset(d_D,0,arraybytes1);
	hipMemset(d_E,0,arraybytes1);
	
		angles<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B,d_A1, d_B1, d_C,d_D,d_E);

      hipMemcpy(h_C, d_C, arraybytes1, hipMemcpyDeviceToHost);
	hipMemcpy(h_D, d_D, arraybytes1, hipMemcpyDeviceToHost);
	hipMemcpy(h_E, d_E, arraybytes1, hipMemcpyDeviceToHost);
	
	for(int i=0; i<720*xx; i++)
	{	result[i%720]+= h_C[i];result_r[i%720]+=h_D[i];result_s[i%720]+=h_E[i];} 

		
	for(int i=0;i<720;i++)
		final[i]=(float) ((float)(result_r[i]-2*result[i]+result_s[i])/(float) result_s[i]); 
	
	end = clock();
     cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
	
	double brk=0; 
	printf("%f\n", cpu_time_used); 
	
	for(int i=0; i<720; i++)
		{
		 
		printf("%f ", final[i]);
	}
	
		/*for(int i=0; i<720; i++)
			brk+=(double) result[i]; 
	printf("%f\n ", brk);
	brk=0;
	for(int i=0; i<720; i++)
			brk+=result_s[i]; 
	printf("%f\n ", brk);
	brk=0;
	for(int i=0; i<720; i++)
			brk+=result_r[i]; 
	//printf("%f\n ", brk);
	brk=0;
		//{printf("%f ", final[i]);   }
	/*printf("\n druga\n " ) ; 
	for(int i=0; i<720; i++)
		{printf("%d ", result_r[i]);   }
	printf("\n treca\n " ) ; 
	for(int i=0; i<720; i++)
		{printf("%d ", result_s[i]);   }*/
	

hipFree(d_A); hipFree(d_B); hipFree(d_C);
	hipFree(h_A); hipFree(h_B); hipFree(h_C);hipFree(d_D);hipFree(h_D);hipFree(d_E);hipFree(h_E);
	hipFree(d_A1);hipFree(h_A1);hipFree(d_B1);hipFree(h_B1);

	
}








    
  
