
#include<stdio.h>
#include<string.h>   
#include<stdlib.h>
#include<math.h>
#include <vector>
#include "hip/hip_vector_types.h"
#include<unistd.h>
#include <time.h>
#include "hip/device_functions.h"
#include "hip/hip_runtime.h"

#include <time.h>
#define fix1 3.14/(60*180)


using namespace std;

__global__ void angles(volatile float *a0, volatile float *b0, volatile float *a1, volatile float*b1, volatile int *hist, volatile int* hist_r, volatile int* hist_s)

{
	int idx= blockIdx.x * blockDim.x + threadIdx.x; 
	
	float ac, bb0,sb1,sb0,cb0,k,bb1,ssb1,cb1,ccb1; 
    int angle;  float fix2=57;
    	bb0=b0[idx];  bb1=b1[idx]; ssb1=sin(bb1); 
     sb0=sin(bb0); cb0=cos(bb0); cb1=cos(bb1); 
    __shared__ int mn[720], r[720], s[720];
   if(threadIdx.x==0 )
    {
        for (int i=0;i<720;i++)
	{ mn[i] = 0; r[i]=0;s[i]=0;} 
    }
    __syncthreads();


   if ( idx<100000)
    {
      
        for(int i=0; i<100000; i++)
        	{
		k=b1[i];
		sb1=k-k*k*k/6 + k*k*k*k*k/120- k*k*k*k*k*k*k/5040+k*k*k*k*k*k*k*k*k/362880-k*k*k*k*k*k*k*k*k*k*k/39916800;
		k=b1[i];
		ccb1=1-k*k/2+k*k*k*k/24-k*k*k*k*k*k/720+k*k*k*k*k*k*k*k/40320-k*k*k*k*k*k*k*k*k*k/3628800;
           ac= acosf(sb0*sb1+ cb0*ccb1*cos((a1[i]-a0[idx])));
		ac= (ac*fix2/0.25); 
	
		angle=(int) ac; 
		  atomicAdd(&mn[angle],1);
		
		}
		
	   for(int i=idx+1; i<100000;i++)
	    { 		
		   k=b0[i]; 
		   sb1=k-k*k*k/6 + k*k*k*k*k/120- k*k*k*k*k*k*k/5040+k*k*k*k*k*k*k*k*k/362880-k*k*k*k*k*k*k*k*k*k*k/39916800;
		   ccb1=1-k*k/2+k*k*k*k/24-k*k*k*k*k*k/720+k*k*k*k*k*k*k*k/40320-k*k*k*k*k*k*k*k*k*k/3628800;
		   ac= acosf(sb0*sb1+ cb0*ccb1*cos((a0[i]-a0[idx])));
	    ac= (ac*fix2/0.25); 
            angle=(int) ac; 
            atomicAdd(&r[angle],2);
	     	   k=b1[i];
		   sb1=k-k*k*k/6 + k*k*k*k*k/120- k*k*k*k*k*k*k/5040+k*k*k*k*k*k*k*k*k/362880-k*k*k*k*k*k*k*k*k*k*k/39916800;
		   ccb1=1-k*k/2+k*k*k*k/24-k*k*k*k*k*k/720+k*k*k*k*k*k*k*k/40320-k*k*k*k*k*k*k*k*k*k/3628800;
          ac= acosf((ssb1*sb1)+ cb1*ccb1*cos((a1[idx]-a1[i])));
            ac= (ac*fix2/0.25); 
	    angle=(int) ac; 
            atomicAdd(&s[angle],2);

                }
	
           }
	

    __syncthreads();

      if(threadIdx.x==0)
    {
        for(int i=0;i<720;i++)
	{  
		
		hist[i+(blockIdx.x*720)]=mn[i]; hist_r[i+(blockIdx.x*720)]=r[i]; hist_s[i+(blockIdx.x*720)]=s[i];}
    }

}

int main(int argc, char *argv[])
{
FILE *real_g; FILE *synthetic_g;
int galaxies_r, galaxies_s; 
clock_t start, end;
start = clock();
	real_g = fopen("data_100k_arcmin.txt","r");
    	synthetic_g = fopen("flat_100k_arcmin.txt","r");	
	fscanf(real_g, "%d", &galaxies_r);
	fscanf(synthetic_g,  "%d", &galaxies_s);
	

int N =100000;
	int xx=782; 
size_t arraybytes = N * sizeof(float);
	size_t arraybytes1 =xx *720 *sizeof(int);
	size_t l=720*sizeof(int);
	size_t l1=720*sizeof(float);
// Allocate input vectors h_A and h_B in host memory
float* h_A = (float*)malloc(arraybytes);
float* h_B = (float*)malloc(arraybytes);
float* h_A1 = (float*)malloc(arraybytes);
float* h_B1 = (float*)malloc(arraybytes);
int* h_C = (int*)malloc(arraybytes1); 
int* h_D = (int*)malloc(arraybytes1); 	
int* h_E = (int*)malloc(arraybytes1); 	
	int* result=(int*)malloc(l); 
	int* result_r=(int*)malloc(l); 
	int* result_s=(int*)malloc(l); 
	float* final=(float*)malloc(l1); 
	for(int i=0; i<galaxies_r; i++)
    {
       
        fscanf(real_g, "%e %e", &h_A[i], &h_B[i]);
       fscanf(synthetic_g, "%e %e", &h_A1[i], &h_B1[i]);
	h_A[i]=h_A[i]*fix1; h_A1[i]=h_A1[i]*fix1; h_B[i]=h_B[i]*fix1; h_B1[i]=h_B1[i]*fix1; }
    fclose(real_g);
	 fclose(synthetic_g);	
	
float* d_A; hipMalloc(&d_A, arraybytes);
float* d_B; hipMalloc(&d_B, arraybytes);
float* d_A1; hipMalloc(&d_A1, arraybytes);
float* d_B1; hipMalloc(&d_B1, arraybytes);
int* d_C; hipMalloc(&d_C, arraybytes1);
	int* d_D; hipMalloc(&d_D, arraybytes1);
	int* d_E; hipMalloc(&d_E, arraybytes1);
// Copy arrays from host memory to device memory
hipMemcpy(d_A, h_A, arraybytes, hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B, arraybytes, hipMemcpyHostToDevice);
hipMemcpy(d_A1, h_A1, arraybytes, hipMemcpyHostToDevice);
hipMemcpy(d_B1, h_B1, arraybytes, hipMemcpyHostToDevice);
// Invoke kernel
	dim3 threadsPerBlock(128);
	
 
    dim3 blocksPerGrid(xx); 
     double cpu_time_used;
     
    
    hipMemset(d_C,0,arraybytes1);
	hipMemset(d_D,0,arraybytes1);
	hipMemset(d_E,0,arraybytes1);
	
		angles<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B,d_A1, d_B1, d_C,d_D,d_E);

      hipMemcpy(h_C, d_C, arraybytes1, hipMemcpyDeviceToHost);
	hipMemcpy(h_D, d_D, arraybytes1, hipMemcpyDeviceToHost);
	hipMemcpy(h_E, d_E, arraybytes1, hipMemcpyDeviceToHost);
	
	for(int i=0; i<720*xx; i++)
	{	result[i%720]+= h_C[i];result_r[i%720]+=h_D[i];result_s[i%720]+=h_E[i];} 

	final[0]=(float) ((float)(result_r[0]-2*result[0]+result_s[0]+200000)/(float)(100000+ result_s[0]));	
	for(int i=1;i<720;i++)
		final[i]=(float) ((float)(result_r[i]-2*result[i]+result_s[i])/(float) result_s[i]); 
	
	end = clock();
     cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
	
	double brk=0; 
	printf("%f\n", cpu_time_used); 
	
	for(int i=0; i<720; i++)
		{
		 
		printf( "%f ", final[i]);
	}
	
		
	

hipFree(d_A); hipFree(d_B); hipFree(d_C);
	hipFree(h_A); hipFree(h_B); hipFree(h_C);hipFree(d_D);hipFree(h_D);hipFree(d_E);hipFree(h_E);
	hipFree(d_A1);hipFree(h_A1);hipFree(d_B1);hipFree(h_B1);

	
}








    
  
