#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<unistd.h>
#include <time.h>
#include "hip/device_functions.h"
#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include <time.h>

using namespace std;

__global__ void angles(volatile float *a0, volatile float *b0,   volatile int *hist)

{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // This should range to SUBMATRIX_SIZE

   
    __shared__ int mn[720];
    if(threadIdx.x==0)
    {
        for (int i=0;i<720;i++)
            mn[i] = 0;
    }
    __syncthreads();


    if (idx<10000)
    {
      
        for(int i=0; i<10000; i++)
        {
            angle= (int)(a0[idx]*b0[i]); 
            
                }

                atomicAdd(&mn[angle],1);

            }
        }
    }

    __syncthreads();

    if(threadIdx.x==0)
    {
        for(int i=0;i<720;i++)
            hist[i+(blockIdx.x*720)]=mn[i];
    }

}

int main(int argc, char *argv[])
{
	
 

int N =10000;
	int angle; 
size_t arraybytes = N * sizeof(float);
	size_t arraybytes1 = 20*720 *sizeof(int);
	size_t l=720*sizeof(int);
// Allocate input vectors h_A and h_B in host memory
float* h_A = (float*)malloc(arraybytes);
float* h_B = (float*)malloc(arraybytes);
int* h_C = (int*)malloc(arraybytes1); 
	
	int* result=(int*)malloc(l); 
	
	for(int i=0; i<10000; i++)
	{ h_A[i]=1.0; h_B[i]=1.0;  }
	h_A[0]=5.0; h_B[1] =3.0; 
float* d_A; hipMalloc(&d_A, arraybytes);
float* d_B; hipMalloc(&d_B, arraybytes);
int* d_C; hipMalloc(&d_C, arraybytes1);
	
// Copy arrays from host memory to device memory
hipMemcpy(d_A, h_A, arraybytes, hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B, arraybytes, hipMemcpyHostToDevice);
// Invoke kernel
int NN=100; 
	
	clock_t start, end;
    int threadsPerBlock=512;
    int blocksPerGrid=20; 
	/* dim3 threadsPerBlock(128, 128);
    dim3 blocksPerGrid(1, 1);
        /*if (NN*NN > 512){
            threadsPerBlock.x = 512;
            threadsPerBlock.y = 512;
            blocksPerGrid.x = ceil(double(NN)/double(threadsPerBlock.x));
            blocksPerGrid.y = ceil(double(NN)/double(threadsPerBlock.y));
        }*/
     double cpu_time_used;
     
     start = clock();
    hipMemset(d_C,0,arraybytes1);
	
 	angles<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C);

    hipMemcpy(h_C, d_C, arraybytes, hipMemcpyDeviceToHost);
	
	result[0] = h_C[0] ; 
	result[1]= h_C[1]; 
	result[2]=h_C[3]; 
/*	for(int i=0; i<N*N; i++)
	{	result[0]= h_C[i]; //angle= h_C[i]; result[angle]++; } */

		
	
	
	end = clock();
     cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
	printf("%f\n", cpu_time_used); 
		for(int i=0; i<3; i++)
			//if(result[i]>0)
		printf("%d ", result[i]);   
// Free device memory
hipFree(d_A); hipFree(d_B); hipFree(d_C);
	hipFree(h_A); hipFree(h_B); hipFree(h_C);
// Free host memory ...
	
}








    
  
