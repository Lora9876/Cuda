#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<unistd.h>
#include <time.h>
#include "hip/device_functions.h"
#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include <time.h>

using namespace std;

__global__ void angles(volatile float *a0, volatile float *b0, volatile float *a1, volatile float*b1,   volatile int *hist, volatile int* hist_r, volatile int* hist_s)

{
	float ac;//721? koliko puta ucitavas i gde da mnozis...zasto float
    int angle; float fix1=0.00029074074; float fix2=57.2957795131;
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 

   
    __shared__ int mn[720], r[720], s[720];
    if(threadIdx.x==0)
    {
        for (int i=0;i<720;i++)
	{ mn[i] = 0; r[i]=0;s[i]=0;} 
    }
    __syncthreads();


    if (idx<10000)
    {
      
        for(int i=0; i<10000; i++)
        	{
            ac= acosf((sin(b0[i]*fix1)*sin(b1[i]*fix1))+ cos(b0[i]*fix1)*cos(b1[i]*fix1)*cos((a1[idx]-a0[idx])*fix1));
		angle=(int) (ac*fix2/0.25); 
             atomicAdd(&mn[angle],1);
                }
	for(int i=idx+1; i<10000; i++)
	{
	     angle= (int)(a0[idx]*a0[i]); 
             atomicAdd(&r[angle],1);
	     angle= (int)(b0[idx]*b0[i]); 
             atomicAdd(&s[angle],1);
	
	}
               

            }
        
    

    __syncthreads();

    if(threadIdx.x==0)
    {
        for(int i=0;i<720;i++)
	{ hist[i+(blockIdx.x*720)]=mn[i]; hist_r[i+(blockIdx.x*720)]=r[i]; hist_s[i+(blockIdx.x*720)]=s[i];}
    }

}

int main(int argc, char *argv[])
{
	
 

int N =10000;
size_t arraybytes = N * sizeof(float);
	size_t arraybytes1 = 20*720 *sizeof(int);
	size_t l=720*sizeof(int);
// Allocate input vectors h_A and h_B in host memory
float* h_A = (float*)malloc(arraybytes);
float* h_B = (float*)malloc(arraybytes);
int* h_C = (int*)malloc(arraybytes1); 
int* h_D = (int*)malloc(arraybytes1); 	
int* h_E = (int*)malloc(arraybytes1); 	
	int* result=(int*)malloc(l); 
	int* result_r=(int*)malloc(l); 
	int* result_s=(int*)malloc(l); 
	for(int i=0; i<10000; i++)
	{ h_A[i]=1.0; h_B[i]=1.0;  }
	h_A[0]=5.0; h_B[1] =3.0; 
float* d_A; hipMalloc(&d_A, arraybytes);
float* d_B; hipMalloc(&d_B, arraybytes);
int* d_C; hipMalloc(&d_C, arraybytes1);
	int* d_D; hipMalloc(&d_D, arraybytes1);
	int* d_E; hipMalloc(&d_E, arraybytes1);
// Copy arrays from host memory to device memory
hipMemcpy(d_A, h_A, arraybytes, hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B, arraybytes, hipMemcpyHostToDevice);
// Invoke kernel
	
	clock_t start, end;
    int threadsPerBlock=736;
    int blocksPerGrid=15; 
     double cpu_time_used;
     
     start = clock();
    hipMemset(d_C,0,arraybytes1);
	hipMemset(d_D,0,arraybytes1);
	hipMemset(d_E,0,arraybytes1);
	
 	angles<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C,d_D,d_E);

      hipMemcpy(h_C, d_C, arraybytes1, hipMemcpyDeviceToHost);
	hipMemcpy(h_D, d_D, arraybytes1, hipMemcpyDeviceToHost);
	hipMemcpy(h_E, d_E, arraybytes1, hipMemcpyDeviceToHost);
	
	for(int i=0; i<720*20; i++)
	{	result[i%720]+= h_C[i];result_r[i%720]+=h_D[i];result_s[i%720]+=h_E[i];} 

		
	
	
	end = clock();
     cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
	printf("%f\n", cpu_time_used); 
		for(int i=0; i<720; i++)
		{printf("%d ", result[i]);   }
	printf("\n druga\n " ) ; 
	for(int i=0; i<720; i++)
		{printf("%d ", result_r[i]);   }
	printf("\n treca\n " ) ; 
	for(int i=0; i<720; i++)
		{printf("%d ", result_s[i]);   }

hipFree(d_A); hipFree(d_B); hipFree(d_C);
	hipFree(h_A); hipFree(h_B); hipFree(h_C);hipFree(d_D);hipFree(h_D);hipFree(d_E);hipFree(h_E);

	
}








    
  
