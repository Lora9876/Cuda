#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<unistd.h>

#include<hip/hip_runtime.h>

using namespace std; 
//const int N = 16; 
//const int blocksize = 16; 
//const int  SUBMATRIX_SIZE=16384 ;
const int thread= 256; 
const int bins=720; 
__global__ void angles(volatile float *a0, volatile float *b0, volatile float *a1, volatile float *b1, int xind, int yind, int max_x, int max_y, volatile int *histi)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x; // ovo proveri
	float angle; 
    idx += xind;
 __shared__ unsigned int shared[bins];
    // za prvu petlju ocistis uvek
    if(threadIdx.x==0)
    {
        for (int i=0;i<bins;i++)
            shared[i] = 0;
    }
    __syncthreads();

	 if (idx<max_x)
    {
       
      //provera

        for(int i=0; i<max_y; i++)
        {
           
                
		
                angle = acos(sin(b0[idx]) *sin(b1[i]) + cos(b0[idx]) * cos(b1[i]) * cos(a0[idx]-a1[0]));
		
		shared[int(angle)]++ ;
		//how to put angle
             //  int *p= &shared[angle];
		//atomicAdd(&shared[int(angle)],1); 
	
	}
	
	
	

          

    __syncthreads();

    if(threadIdx.x==0)
    {
        for(int i=0;i<bins;i++)
            histi[i]=shared[i];
    }



}
}

 void read_the_files()
{
	//reading files 1 and 2 
	FILE *real_g; FILE *synthetic_g;
	int galaxies_r, galaxies_s; 
	float *a0, *a1, *b0, *b1;
	real_g = fopen("data_100k_arcmin.txt","r");
    	synthetic_g = fopen("flat_100k_arcmin.txt","r");	
	 fscanf(real_g, "%d", &galaxies_r);
	 fscanf(synthetic_g,  "%d", &galaxies_s);
	
	
	a0= (float*) malloc(galaxies_r* sizeof(float));
	b0= (float*) malloc(galaxies_r* sizeof(float)); 
	a1= (float*) malloc(galaxies_s* sizeof(float)); 
	b1= (float*) malloc(galaxies_s* sizeof(float)); 
	for(int i=0; i<galaxies_r; i++)
    {
       
        fscanf(real_g, "%f %f", &a0[i], &b0[i]);
       fscanf(synthetic_g, "%f %f", &a1[i], &b1[i]);
    }	
	 for(int i=0; i<galaxies_r; i++)
	 {
	 	printf("%d", galaxies_r); 
		 printf("%d", galaxies_s); 
		
	 
	 }
//for(int i=0; i<galaxies_r; i++) printf("%f", a0[i]);
	 
    dim3 grid, block;
    
    grid.x = thread; 
    block.x = 1024; 
	 float *aa1, *bb1, *aa0, *bb0; 
	 
    hipMalloc((void **) &aa0, galaxies_r* sizeof(float));
    hipMalloc((void **) &bb0, galaxies_r* sizeof(float));

    hipMalloc((void **) &aa1, galaxies_s* sizeof(float));
    hipMalloc((void **) &bb1, galaxies_s* sizeof(float) );

    // dovoljno memorije?
    

    // Initialize array to all 0's
    hipMemset(aa0,0,galaxies_r* sizeof(float));
    hipMemset(bb0,0,galaxies_r* sizeof(float));
    hipMemset(aa1,0,galaxies_s* sizeof(float));
    hipMemset(bb1,0,galaxies_s* sizeof(float));

    hipMemcpy(aa0, a0, galaxies_r* sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(bb0, b0,galaxies_r* sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(aa1, a1, galaxies_s* sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(bb1, b1,galaxies_s* sizeof(float), hipMemcpyHostToDevice );

    int x, y;

	 /*  int num_x = galaxies_r/ SUBMATRIX_SIZE;
    int num_y = galaxies_s / SUBMATRIX_SIZE;
    
    // Take care of edges of matrix.
    if (galaxies_r%SUBMATRIX_SIZE != 0)
    {
        num_x ++;
    }
    if (galaxies_s%SUBMATRIX_SIZE != 0)
    {
        num_y ++;
     */
	 
	 //preparing the histogram array 
	 int *hist, *histi , *tmp;

   
    int size_h_bytes = 720*sizeof(int);

    hist = (int*)malloc(size_h_bytes);
    memset(hist, 0, size_h_bytes);

   
    hipMalloc((void **) &histi, (size_h_bytes));
    hipMemset(histi, 0, size_h_bytes);

    unsigned long  *hist_array;

    int hist_array_size = 720 * sizeof(unsigned long);
    hist_array =  (unsigned long*)malloc(hist_array_size);
  
    memset(hist_array,0,hist_array_size); 
	 hipMemset(tmp,0,size_h_bytes);
	 
	   angles<<<grid,block>>>(aa0, bb0, aa1, bb1, 0, 0, 512, 512, tmp);
            hipMemcpy(hist, tmp, size_h_bytes, hipMemcpyDeviceToHost);
	 
	 for(int i=0; i<720; i++)
		printf("%d", hist[i]);
 }
 //prepration for the kernel
	 


int main()
{
	float alpha1= 4646.98;
	float b1= 3749.51;
	float a2=4644.35; 
	float b2=3749.52;
	
	float theta1= acos(sin(b1)*sin(b2) + cos(b1)*cos(b2) *cos(alpha1-a2));
	
	printf("%f\n", b1);
	printf("%f\n", theta1);
	
 	read_the_files(); 
	
	return EXIT_SUCCESS;
}

