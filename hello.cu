#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<unistd.h>
#include <time.h>

#include<hip/hip_runtime.h>
#define PI 3.14159265
const int val= 180.0 / PI;
using namespace std; 
//const int N = 16; 
//const int blocksize = 16; 
//const int  SUBMATRIX_SIZE=16384 ;
const int thread= 256; 
const float fix =1/60 ; 
const int bins=720; 
#define arccos[180]={170.565836,167.031761,164.312253,162.013086,159.981700,158.139924,156.441357,154.855789,153.362253,151.945493,150.594000,149.298836,148.052901,146.850438,145.686705,144.557743,143.460201,142.391218,141.348325,140.329374,139.332483,138.355992,137.398427,136.458473,135.534951,134.626797,133.733048,132.852831,131.985347,131.129867,130.285721,129.452292,128.629010,127.815347,127.010813,126.214954,125.427343,124.647584,123.875305,123.110156,122.351810,121.599957,120.854305,120.114580,119.380521,118.651879,117.928421,117.209924,116.496173,115.786968,115.082114,114.381427,113.684729,112.991851,112.302629,111.616908,110.934536,110.255369,109.579267,108.906094,108.235721,107.568021,106.902872,106.240155,105.579755,104.921560,104.265462,103.611353,102.959132,102.308697,101.659949,101.012792,100.367132,99.722875,99.079931,98.438211,97.797626,97.158091,96.519520,95.881828,95.244934,94.608753,93.973206,93.338211,92.703689,92.069559,91.435743,90.802162,90.168737,89.535391,88.902044,88.268620,87.635039,87.001223,86.367093,85.732570,85.097575,84.462028,83.825848,83.188953,82.551261,81.912690,81.273155,80.632570,79.990850,79.347906,78.703650,78.057989,77.410832,76.762084,76.111649,75.459428,74.805320,74.149221,73.491026,72.830626,72.167909,71.502760,70.835060,70.164687,69.491514,68.815412,68.136245,67.453873,66.768152,66.078930,65.386052,64.689354,63.988667,63.283813,62.574608,61.860858,61.142360,60.418902,59.690260,58.956201,58.216476,57.470825,56.718972,55.960625,55.195477,54.423197,53.643438,52.855827,52.059968,51.255435,50.441772,49.618489,48.785060,47.940914,47.085434,46.217951,45.337733,44.443985,43.535830,42.612308,41.672354,40.714790,39.738299,38.741408,37.722457,36.679563,35.610580,34.513038,33.384076,32.220343,31.017880,29.771945,28.476782,27.125288,25.708528,24.214992,22.629425,20.930857,19.089081,17.057696,14.758528,12.039020,8.504946,0.000005};
__global__ void angles(volatile float *a0, volatile float *b0, volatile float *a1, volatile float *b1, int xind, int yind, int max_x, int max_y, volatile int *histi)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;// ovo proveri
	float angle; 
   	idx+=xind; 
	
	
 	__shared__ unsigned int shared[bins];
    		// za prvu petlju ocistis uvek
    			if(threadIdx.x==0)
   			 {
       			 for (int i=0;i<bins;i++)
         		   shared[i] = 0;
   			 }
	
   	 __syncthreads();

	
      //provera
			for(int i=yind; i<max_y; i++)
       			
			{ angle = sin(b0[idx]*fix) *sin(b1[i]*fix) + cos(b0[idx]*fix) * cos(b1[i]*fix) * cos(fix*a0[idx]*-fix*a1[i]);
				shared[int(arccos[angle])]++; 	
				
			//nadji nacin da atomic add proradi :D
			//atomicAdd(&shared[int(angle)],1); 
				}
   	 __syncthreads();

    if(threadIdx.x==0)
    {
        for(int i=0;i<bins;i++)
            histi[i]=shared[i];
    }



}


 void read_the_files()
{
	//reading files 1 and 2 
	FILE *real_g; FILE *synthetic_g;
	int galaxies_r, galaxies_s; 
	float *a0, *a1, *b0, *b1;
	real_g = fopen("data_100k_arcmin.txt","r");
    	synthetic_g = fopen("flat_100k_arcmin.txt","r");	
	 fscanf(real_g, "%d", &galaxies_r);
	 fscanf(synthetic_g,  "%d", &galaxies_s);
	
	
	a0= (float*) malloc(galaxies_r* sizeof(float));
	b0= (float*) malloc(galaxies_r* sizeof(float)); 
	a1= (float*) malloc(galaxies_s* sizeof(float)); 
	b1= (float*) malloc(galaxies_s* sizeof(float)); 
	for(int i=0; i<galaxies_r; i++)
    {
       
        fscanf(real_g, "%e %e", &a0[i], &b0[i]);
       fscanf(synthetic_g, "%e %e", &a1[i], &b1[i]);
    }	
	/*for(int i=0; i<galaxies_r; i++)
	{
		a0[i]=a1[i]=b0[i]=b1[i]=0.5; 
	
	}*/
	 fclose(real_g);
	 fclose(synthetic_g);
//for(int i=0; i<galaxies_r; i++) printf("%f", a0[i]);
	 
    dim3 grid, block;
    
    grid.x = 1024; 
	 grid.y=1024;
	// grid.y=1024; 
    block.x = 1; 
	 float *aa1, *bb1, *aa0, *bb0; 
	 
    hipMalloc((void **) &aa0, galaxies_r* sizeof(float));
    hipMalloc((void **) &bb0, galaxies_r* sizeof(float));

    hipMalloc((void **) &aa1, galaxies_s* sizeof(float));
    hipMalloc((void **) &bb1, galaxies_s* sizeof(float) );

    // dovoljno memorije?
    

    // Initialize array to all 0's
    hipMemset(aa0,0,galaxies_r* sizeof(float));
    hipMemset(bb0,0,galaxies_r* sizeof(float));
    hipMemset(aa1,0,galaxies_s* sizeof(float));
    hipMemset(bb1,0,galaxies_s* sizeof(float));

    hipMemcpy(aa0, a0, galaxies_r* sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(bb0, b0,galaxies_r* sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(aa1, a1, galaxies_s* sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(bb1, b1,galaxies_s* sizeof(float), hipMemcpyHostToDevice );

    int x, y;

	 /*  int num_x = galaxies_r/ SUBMATRIX_SIZE;
    int num_y = galaxies_s / SUBMATRIX_SIZE;
    
    // Take care of edges of matrix.
    if (galaxies_r%SUBMATRIX_SIZE != 0)
    {
        num_x ++;
    }
    if (galaxies_s%SUBMATRIX_SIZE != 0)
    {
        num_y ++;
     */
	 
	 //preparing the histogram array 
	 int *hist, *histi , *tmp; 
	 
   
    int size_h_bytes = 720*sizeof(int);

    hist = (int*)malloc(size_h_bytes);
    memset(hist, 0, size_h_bytes);

   
    hipMalloc((void **) &tmp, (size_h_bytes));
    hipMemset(tmp, 0, size_h_bytes);

    unsigned long  *hist_array;

    int hist_array_size = 720 * sizeof(unsigned long);
    hist_array =  (unsigned long*)malloc(hist_array_size);
  
    memset(hist_array,0,hist_array_size); 
	 hipMemset(tmp, 0,size_h_bytes);
	 
	   angles<<<grid,block>>>(aa0, bb0, aa1, bb1, 0, 0, 512, 512, tmp);
            hipMemcpy(hist, tmp, size_h_bytes, hipMemcpyDeviceToHost);
	 
	 for(int i=0; i<720; i++)
		printf("%d ", hist[i]);
	 
	 
    free(a1);
    free(b1);
    free(a0);
    free(b0);

    hipFree(aa1);
    hipFree(aa0);  
    hipFree(bb0);
    hipFree(bb1);  
    hipFree(tmp);

 }
 //prepration for the kernel
	 


int main()
{
	float alpha1= 4646.98;
	float b1= 3749.51;
	float a2=4644.35; 
	float b2=3749.52;
	
	float theta1= acos(sin(b1)*sin(b2) + cos(b1)*cos(b2) *cos(alpha1-a2));
	
	printf("%f\n", b1);
	printf("%f\n", theta1);
	
	
	 clock_t start, end;
     double cpu_time_used;
     
     start = clock();
    
 
		
 	//read_the_files(); 
	// do some calculations
 	end = clock();
     cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
	printf("%f", cpu_time_used); 
	return EXIT_SUCCESS;
}


