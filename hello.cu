
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>  
const int N = 16; 
const int blocksize = 16; 
const int  SUBMATRIX_SIZE=16384 ;
const int thread= 256; 
 void read_the_files()
{
	//reading files 1 and 2 
	FILE *real_g; FILE *synthetic_g;
	int galaxies_r, galaxies_s; 
	float *a0, *a1, *b0, *b1;
	real_g = fopen("data_100k_arcmin.txt","r");
    	synthetic_g = fopen("flat_100k_arcmin.txt","r");	
	 fscanf(real_g, "%d", &galaxies_r);
	 fscanf(synthetic_g,  "%d", &galaxies_s);
	
	
	a0= (float*) malloc(galaxies_r* sizeof(float));
	b0= (float*) malloc(galaxies_r* sizeof(float)); 
	a1= (float*) malloc(galaxies_s* sizeof(float)); 
	b1= (float*) malloc(galaxies_s* sizeof(float)); 
	for(int i=0; i<galaxies_r; i++)
    {
       
        fscanf(real_g, "%f %f", &a0[i], &b0[i]);
       fscanf(synthetic_g, "%f %f", &a1[i], &b1[i]);
    }		    
//for(int i=0; i<galaxies_r; i++) printf("%f", a0[i]); 
	 
    dim3 grid, block;
    
    grid.x = 8192/thread; 
    block.x = SUBMATRIX_SIZE/grid.x; 
	 float *aa1, *bb1, *aa0, *bb0; 
	 
    hipMalloc((void **) &aa0, galaxies_r* sizeof(float));
    hipMalloc((void **) &bb0, galaxies_r* sizeof(float));

    hipMalloc((void **) &aa1, galaxies_s* sizeof(float));
    hipMalloc((void **) &bb1, galaxies_s* sizeof(float) );

    // dovoljno memorije?
    

    // Initialize array to all 0's
    hipMemset(aa0,0,galaxies_r* sizeof(float));
    hipMemset(bb0,0,galaxies_r* sizeof(float));
    hipMemset(aa1,0,galaxies_s* sizeof(float));
    hipMemset(bb1,0,galaxies_s* sizeof(float));

    hipMemcpy(aa0, a0, galaxies_r* sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(bb0, b0,galaxies_r* sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(aa1, a1, galaxies_s* sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(bb1, b1,galaxies_s* sizeof(float), hipMemcpyHostToDevice );

    int x, y;

    int num_x = galaxies_r/ SUBMATRIX_SIZE;
    int num_y = galaxies_s / SUBMATRIX_SIZE;
    
    // Take care of edges of matrix.
    if (galaxies_r%SUBMATRIX_SIZE != 0)
    {
        num_x ++;
    }
    if (galaxies_s%SUBMATRIX_SIZE != 0)
    {
        num_y ++;
    }

	 //preparing the histogram array 
	 int *hist, *dev_h;

    int size_h = SUBMATRIX_SIZE * thread;
    int size_h_bytes = size_h*sizeof(int);

    hist = (int*)malloc(size_h_bytes);
    memset(hist, 0, size_h_bytes);

   
    hipMalloc((void **) &dev_h, (size_h_bytes));
    hipMemset(dev_h, 0, size_h_bytes);

    unsigned long  *hist_array;

    int hist_array_size = thread * sizeof(unsigned long);
    hist_array =  (unsigned long*)malloc(hist_array_size);
    printf("Size of histogram array: %d bytes\n",hist_array_size);
    memset(hist_array,0,hist_array_size); 
	 
 //prepration for the kernel
	 
}
//__global__ 
/*void hello(char *a, char *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
}
 */
int main()
{
	float alpha1= 4646.98;
	float b1= 3749.51;
	float a2=4644.35; 
	float b2=3749.52;
	
	float theta1= acos(sin(b1)*sin(b2) + cos(b1)*cos(b2) *cos(alpha1-a2));
	
	printf("%f\n", b1);
	printf("%f\n", theta1);
	
 	read_the_files(); 
	/*char *ad;
	int *bd;
	const char csize = N*sizeof(char);
	const char isize = N*sizeof(char);
 
	printf("%s", a);
 
	cudaMalloc( (void**)&ad, csize ); 
	cudaMalloc( (void**)&bd, isize ); 
	cudaMemcpy( ad, a, csize, cudaMemcpyHostToDevice ); 
	cudaMemcpy( bd, b, isize, cudaMemcpyHostToDevice ); 
	
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	cudaMemcpy( a, ad, csize, cudaMemcpyDeviceToHost ); 
	cudaFree( ad );
	cudaFree( bd );
	
	printf("%s\n", a);*/
	return EXIT_SUCCESS;
}


