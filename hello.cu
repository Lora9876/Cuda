#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>  
const int N = 16; 
const int blocksize = 16; 
 
//__global__ 
/*void hello(char *a, char *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
}
 */
int main()
{
	float alpha1= 4646.98;
	float b1= 3749.51;
	float a2=4644.35; 
	float b2=3749.52;
	
	float theta1= acos(sin(b1)*sin(b2) + cos(b1)*cos(b2) *cos(alpha1-a2));
	
	printf("%f\n", b1);
	printf("%f\n", theta1);
	
 
	/*char *ad;
	int *bd;
	const char csize = N*sizeof(char);
	const char isize = N*sizeof(char);
 
	printf("%s", a);
 
	hipMalloc( (void**)&ad, csize ); 
	hipMalloc( (void**)&bd, isize ); 
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
	
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
	hipFree( ad );
	hipFree( bd );
	
	printf("%s\n", a);*/
	return EXIT_SUCCESS;
}

void read_the_files(FILE *real_g, FILE *synthetic_g)
{
	int galaxies_r, galaxies_s; 
	float *a0, *a1, *b0, *b1;
	real_g = fopen("data_100k_arcmin.txt","r");
    synthetic_g = fopen("flat_100k_arcmin.txt","r");	
	 fscanf(real_g, "%d", &galaxies_r);
	 fsancf(synthetic_g,  "%d", &galaxies_s);
	int size_of_galaxy_array0 = NUM_GALAXIES0 * sizeof(float);
	
	a0= (float*) malloc(galaxies_r* sizeof(float);
	b0= (float*) malloc(galaxies_r* sizeof(float); 
	a1= (float*) malloc(galaxies_s* sizeof(float); 
	b1= (float*) malloc(galaxies_s* sizeof(float); 
	for(int i=0; i<galaxies_r; i++)
    {
       
        fscanf(real_g, "%f %f", &a0[i], &b0[i]);
       fscanf(synthetic_g, "%f %f", &a1[i], &b1[i]);
    }		    
for(int i=0; i<galaxies_r; i++) printf("%f", a0[i]); 
}
