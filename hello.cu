#include "hip/hip_runtime.h"
#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<unistd.h>
#include <time.h>

#include<hip/hip_runtime.h>
#define PI 3.14159265
const int val= 180.0 / PI;
using namespace std; 
//const int N = 16; 
//const int blocksize = 16; 
//const int  SUBMATRIX_SIZE=16384 ;
const int thread= 256; 
const int bins=720; 
double arccos(x) {
   return (-0.69813170079773212 * x * x - 0.87266462599716477) * x + 1.5707963267948966;
}
__global__ void angles(volatile float *a0, volatile float *b0, volatile float *a1, volatile float *b1, int xind, int yind, int max_x, int max_y, volatile int *histi)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;// ovo proveri
	float angle; 
   	idx+=xind; 
	
	
 	__shared__ unsigned int shared[bins];
    		// za prvu petlju ocistis uvek
    			if(threadIdx.x==0)
   			 {
       			 for (int i=0;i<bins;i++)
         		   shared[i] = 0;
   			 }
	
   	 __syncthreads();

	
      //provera
			for(int i=yind; i<max_y; i++)
       	
			{ angle = arccos(sin(b0[idx]) *sin(b1[i]) + cos(b0[idx]) * cos(b1[i]) * cos(a0[idx]-a1[i]));
		
				shared[int(angle/0.25)]++ ;
			//nadji nacin da atomic add proradi :D
			//atomicAdd(&shared[int(angle)],1); 
				}
   	 __syncthreads();

    if(threadIdx.x==0)
    {
        for(int i=0;i<bins;i++)
            histi[i]=shared[i];
    }



}


 void read_the_files()
{
	//reading files 1 and 2 
	FILE *real_g; FILE *synthetic_g;
	int galaxies_r, galaxies_s; 
	float *a0, *a1, *b0, *b1;
	real_g = fopen("data_100k_arcmin.txt","r");
    	synthetic_g = fopen("flat_100k_arcmin.txt","r");	
	 fscanf(real_g, "%d", &galaxies_r);
	 fscanf(synthetic_g,  "%d", &galaxies_s);
	
	
	a0= (float*) malloc(galaxies_r* sizeof(float));
	b0= (float*) malloc(galaxies_r* sizeof(float)); 
	a1= (float*) malloc(galaxies_s* sizeof(float)); 
	b1= (float*) malloc(galaxies_s* sizeof(float)); 
	for(int i=0; i<galaxies_r; i++)
    {
       
        fscanf(real_g, "%e %e", &a0[i], &b0[i]);
       fscanf(synthetic_g, "%e %e", &a1[i], &b1[i]);
    }	
	/*for(int i=0; i<galaxies_r; i++)
	{
		a0[i]=a1[i]=b0[i]=b1[i]=0.5; 
	
	}*/
	 fclose(real_g);
	 fclose(synthetic_g);
//for(int i=0; i<galaxies_r; i++) printf("%f", a0[i]);
	 
    dim3 grid, block;
    
    grid.x = 1024; 
	 grid.y=1024;
	// grid.y=1024; 
    block.x = 1; 
	 float *aa1, *bb1, *aa0, *bb0; 
	 
    hipMalloc((void **) &aa0, galaxies_r* sizeof(float));
    hipMalloc((void **) &bb0, galaxies_r* sizeof(float));

    hipMalloc((void **) &aa1, galaxies_s* sizeof(float));
    hipMalloc((void **) &bb1, galaxies_s* sizeof(float) );

    // dovoljno memorije?
    

    // Initialize array to all 0's
    hipMemset(aa0,0,galaxies_r* sizeof(float));
    hipMemset(bb0,0,galaxies_r* sizeof(float));
    hipMemset(aa1,0,galaxies_s* sizeof(float));
    hipMemset(bb1,0,galaxies_s* sizeof(float));

    hipMemcpy(aa0, a0, galaxies_r* sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(bb0, b0,galaxies_r* sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(aa1, a1, galaxies_s* sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(bb1, b1,galaxies_s* sizeof(float), hipMemcpyHostToDevice );

    int x, y;

	 /*  int num_x = galaxies_r/ SUBMATRIX_SIZE;
    int num_y = galaxies_s / SUBMATRIX_SIZE;
    
    // Take care of edges of matrix.
    if (galaxies_r%SUBMATRIX_SIZE != 0)
    {
        num_x ++;
    }
    if (galaxies_s%SUBMATRIX_SIZE != 0)
    {
        num_y ++;
     */
	 
	 //preparing the histogram array 
	 int *hist, *histi , *tmp; 
	 
   
    int size_h_bytes = 720*sizeof(int);

    hist = (int*)malloc(size_h_bytes);
    memset(hist, 0, size_h_bytes);

   
    hipMalloc((void **) &tmp, (size_h_bytes));
    hipMemset(tmp, 0, size_h_bytes);

    unsigned long  *hist_array;

    int hist_array_size = 720 * sizeof(unsigned long);
    hist_array =  (unsigned long*)malloc(hist_array_size);
  
    memset(hist_array,0,hist_array_size); 
	 hipMemset(tmp, 0,size_h_bytes);
	 
	   angles<<<grid,block>>>(aa0, bb0, aa1, bb1, 0, 0, 512, 512, tmp);
            hipMemcpy(hist, tmp, size_h_bytes, hipMemcpyDeviceToHost);
	 
	 for(int i=0; i<720; i++)
		printf("%d ", hist[i]);
	 
	 
    free(a1);
    free(b1);
    free(a0);
    free(b0);

    hipFree(aa1);
    hipFree(aa0);  
    hipFree(bb0);
    hipFree(bb1);  
    hipFree(tmp);

 }
 //prepration for the kernel
	 


int main()
{
	float alpha1= 4646.98;
	float b1= 3749.51;
	float a2=4644.35; 
	float b2=3749.52;
	
	float theta1= acos(sin(b1)*sin(b2) + cos(b1)*cos(b2) *cos(alpha1-a2));
	
	printf("%f\n", b1);
	printf("%f\n", theta1);
	
	
	 clock_t start, end;
     double cpu_time_used;
     
     start = clock();
    
    

 	read_the_files(); 
	// do some calculations
 	end = clock();
     cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
	printf("%f", cpu_time_used); 
	return EXIT_SUCCESS;
}


