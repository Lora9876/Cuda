#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <math.h>
#include <unistd.h>
#include <time.h>

#include <hip/hip_runtime.h>

#define PI 3.14159265

const int val = 180.0 / PI;
const int thread = 256;
const float fix = 1 / 60 * PI / 180;
const int bins = 720;

using namespace std;

__global__ void angles(volatile float *a0, volatile float *b0, volatile float *a1, volatile float *b1, int xind, int yind, int max_x, int max_y, volatile int *histi)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	float angle;
	idx += xind;
	
	__shared__ unsigned int shared[bins];

	// Always clear the first iteration
	if (threadIdx.x == 0) {
		for (int i = 0; i < bins; i++) {
			shared[i] = 0;
		}
	}
	
	__syncthreads();

	// Check
	for (int i = yind; i < yind + max_y; i++) {
		angle = acosf((sin(b0[idx]) * sin(b1[i]) + cos(b0[idx]) * cos(b1[i]) * cos(a0[idx] * -a1[i])) * 1 / val);
		shared[int(angle)]++;

		// TODO: Make atomicAddwork :D
		// atomicAdd(&shared[int(angle)], 1);
	}

	__syncthreads();

	if (threadIdx.x == 0) {
		for (int i = 0; i < bins; i++) {
			histi[i] = shared[i];
		}
	}
}

void read_the_files()
{
	// Reading files 1 and 2
	FILE *real_g;
	FILE *synthetic_g;
	int galaxies_r, galaxies_s;
	float *a0, *a1, *b0, *b1;

	real_g = fopen("data_100k_arcmin.txt", "r");
	synthetic_g = fopen("flat_100k_arcmin.txt", "r");
	fscanf(real_g, "%d", &galaxies_r);
	fscanf(synthetic_g, "%d", &galaxies_s);
	
	a0 = (float*) malloc(galaxies_r * sizeof(float));
	b0 = (float*) malloc(galaxies_r * sizeof(float));
	a1 = (float*) malloc(galaxies_s * sizeof(float));
	b1 = (float*) malloc(galaxies_s * sizeof(float));

	for (int i = 0; i < galaxies_r; i++) {
		fscanf(real_g, "%e %e", &a0[i], &b0[i]);
		fscanf(synthetic_g, "%e %e", &a1[i], &b1[i]);

		a0[i] *= fix;
		b0[i] *= fix;
		a1[i] *= fix;
		b1[i] *= fix;
	}

	fclose(real_g);
	fclose(synthetic_g);

	dim3 grid, block;

	grid.x = 1024;
	grid.y = 1024;
	block.x = 1;
	float *aa1, *bb1, *aa0, *bb0;

	hipMalloc((void **) &aa0, galaxies_r * sizeof(float));
	hipMalloc((void **) &bb0, galaxies_r * sizeof(float));

	hipMalloc((void **) &aa1, galaxies_s * sizeof(float));
	hipMalloc((void **) &bb1, galaxies_s * sizeof(float));

	// Do we have enough memory?

	// Initialize array to all 0's
	hipMemset(aa0, 0, galaxies_r * sizeof(float));
	hipMemset(bb0, 0, galaxies_r * sizeof(float));
	hipMemset(aa1, 0, galaxies_s * sizeof(float));
	hipMemset(bb1, 0, galaxies_s * sizeof(float));

	hipMemcpy(aa0, a0, galaxies_r * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(bb0, b0, galaxies_r * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(aa1, a1, galaxies_s * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(bb1, b1, galaxies_s * sizeof(float), hipMemcpyHostToDevice);

	int x, y;

	// Preparing the histogram array
	int *hist, *histi, *tmp;

	int size_h_bytes = 720 * sizeof(int);

	hist = (int*)malloc(size_h_bytes);
	memset(hist, 0, size_h_bytes);

	hipMalloc((void **) &tmp, (size_h_bytes));
	hipMemset(tmp, 0, size_h_bytes);

	unsigned long *hist_array;

	int hist_array_size = 720 * sizeof(unsigned long);
	hist_array = (unsigned long*)malloc(hist_array_size);

	memset(hist_array, 0, hist_array_size);
	hipMemset(tmp, 0, size_h_bytes);

	for (int i = 0; i < 9; i++) {
		y = i * 512 * 512;

		for (int j = 0; j < 9; j++) {
			x = 512 * 512 * i;

			angles<<<grid,block>>>(aa0, bb0, aa1, bb1, x, y, 512, 512, tmp);
			hipMemcpy(hist, tmp, size_h_bytes, hipMemcpyDeviceToHost);

			for(int i = 0; i < 720; i++) {
				hist_array[i] += hist[i];
			}
		}
	}

	for(int i = 0; i < 720; i++) {
		printf("%d ", hist_array[i]);
	}

	free(a1);
	free(b1);
	free(a0);
	free(b0);

	hipFree(aa1);
	hipFree(aa0);
	hipFree(bb0);
	hipFree(bb1);
	hipFree(tmp);
}

// Preparation for the kernel

int main()
{
	float alpha1 = 4646.98;
	float b1 = 3749.51;
	float a2 = 4644.35;
	float b2 = 3749.52;

	float theta1 = acos(sin(b1) * sin(b2) + cos(b1) * cos(b2) * cos(alpha1 - a2));

	printf("%f\n", b1);
	printf("%f\n", theta1);

	clock_t start, end;
	double cpu_time_used;

	start = clock();
	read_the_files();
	// Do some calculations
	end = clock();

	cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
	printf("%f", cpu_time_used);

	return EXIT_SUCCESS;
}
