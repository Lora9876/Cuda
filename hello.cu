#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>  
//const int N = 16; 
//const int blocksize = 16; 
//const int  SUBMATRIX_SIZE=16384 ;
const int thread= 256; 
const int bins=720; 
__global__ void angles(volatile float *a0, volatile float *b0, volatile float *a1, volatile float *b1, int xind, int yind, int max_x, int max_y, volatile int *histi)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x; // ovo proveri
	float angle; 
    idx += xind;
 __shared__ int shared[bins];
    // za prvu petlju ocistis uvek
    if(threadIdx.x==0)
    {
        for (int i=0;i<bins;i++)
            shared[i] = 0;
    }
    __syncthreads();

	 if (idx<max_x)
    {
       
      //provera

        for(int i=yind; i<max_y; i++)
        {
           
                
		
                angle = sin(b0[idx]) *sin(b1[i]) + cos(b0[idx]) * cos(b1[i]) * cos(a0[idx]-a1[0]);
		//how to put angle
               
		atomicAdd(&shared[angle],1); 
                
	}
	
	
	

          

    __syncthreads();

    if(threadIdx.x==0)
    {
        for(int i=0;i<bins;i++)
            histi[i]=shared[i];
    }



}


 void read_the_files()
{
	//reading files 1 and 2 
	FILE *real_g; FILE *synthetic_g;
	int galaxies_r, galaxies_s; 
	float *a0, *a1, *b0, *b1;
	real_g = fopen("data_100k_arcmin.txt","r");
    	synthetic_g = fopen("flat_100k_arcmin.txt","r");	
	 fscanf(real_g, "%d", &galaxies_r);
	 fscanf(synthetic_g,  "%d", &galaxies_s);
	
	
	a0= (float*) malloc(galaxies_r* sizeof(float));
	b0= (float*) malloc(galaxies_r* sizeof(float)); 
	a1= (float*) malloc(galaxies_s* sizeof(float)); 
	b1= (float*) malloc(galaxies_s* sizeof(float)); 
	for(int i=0; i<galaxies_r; i++)
    {
       
        fscanf(real_g, "%f %f", &a0[i], &b0[i]);
       fscanf(synthetic_g, "%f %f", &a1[i], &b1[i]);
    }		    
//for(int i=0; i<galaxies_r; i++) printf("%f", a0[i]);
	 
    dim3 grid, block;
    
    grid.x = thread; 
    block.x = 1024; 
	 float *aa1, *bb1, *aa0, *bb0; 
	 
    hipMalloc((void **) &aa0, galaxies_r* sizeof(float));
    hipMalloc((void **) &bb0, galaxies_r* sizeof(float));

    hipMalloc((void **) &aa1, galaxies_s* sizeof(float));
    hipMalloc((void **) &bb1, galaxies_s* sizeof(float) );

    // dovoljno memorije?
    

    // Initialize array to all 0's
    hipMemset(aa0,0,galaxies_r* sizeof(float));
    hipMemset(bb0,0,galaxies_r* sizeof(float));
    hipMemset(aa1,0,galaxies_s* sizeof(float));
    hipMemset(bb1,0,galaxies_s* sizeof(float));

    hipMemcpy(aa0, a0, galaxies_r* sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(bb0, b0,galaxies_r* sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(aa1, a1, galaxies_s* sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy(bb1, b1,galaxies_s* sizeof(float), hipMemcpyHostToDevice );

    int x, y;

    int num_x = galaxies_r/ SUBMATRIX_SIZE;
    int num_y = galaxies_s / SUBMATRIX_SIZE;
    
    // Take care of edges of matrix.
    if (galaxies_r%SUBMATRIX_SIZE != 0)
    {
        num_x ++;
    }
    if (galaxies_s%SUBMATRIX_SIZE != 0)
    {
        num_y ++;
    }

	 //preparing the histogram array 
	 int *hist, *dev_h;

   
    int size_h_bytes = 720*sizeof(int);

    hist = (int*)malloc(size_h_bytes);
    memset(hist, 0, size_h_bytes);

   
    hipMalloc((void **) &histi, (size_h_bytes));
    hipMemset(histi, 0, size_h_bytes);

    unsigned long  *hist_array;

    int hist_array_size = 720 * sizeof(unsigned long);
    hist_array =  (unsigned long*)malloc(hist_array_size);
  
    memset(hist_array,0,hist_array_size); 
	 hipMemset(tmp,0,size_hist_bytes);
	   angles<<<grid,block>>>(aa0, bb00,aa1, bb1, 0, 0, 512, 512, tmp);
            hipMemcpy(hist, tmp, size_h_bytes, hipMemcpyDeviceToHost);
	 
	 for(int i=0; i<720; i++)
		printf("%d", hist[i]);
 }
 //prepration for the kernel
	 
}
//__global__ 
/*void hello(char *a, char *b) 
{
	a[threadIdx.x] += b[threadIdx.x];
}
 */
void getDeviceDiagnostics(int tot_gals, int n_coords){

 ////////////////////////////////////////////////////////////////////////////
    // Now get the info from the device.
    ////////////////////////////////////////////////////////////////////////////
   
        printf("\n------ CUDA device diagnostics ------\n\n");

        
        int nx = SUBMATRIX_SIZE;
        int ncalc = nx * nx;
        int gpu_mem_needed = int(tot_gals * sizeof(float)) * n_coords; // need to allocate ra, dec.
        printf("Requirements: %d calculations and %d bytes memory on the GPU \n\n", ncalc, gpu_mem_needed);

        int deviceCount = 0;
        hipError_t error_id = hipGetDeviceCount(&deviceCount);
        if (error_id != hipSuccess) {
            printf( "hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id) );
        }
        // This function call returns 0 if there are no CUDA capable devices.
        if (deviceCount == 0)
            printf("There is no device supporting CUDA\n");
        else
            printf("Found %d CUDA Capable device(s)\n", deviceCount);


        int dev=0;
        for (dev = 0; dev < deviceCount; ++dev) {
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, dev);
            printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

            printf("  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
                    (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);


            printf("  Warp size:                                     %d\n", deviceProp.warpSize);
            printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
            printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
                    deviceProp.maxThreadsDim[0],
                    deviceProp.maxThreadsDim[1],
                    deviceProp.maxThreadsDim[2]);
            printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
                    deviceProp.maxGridSize[0],
                    deviceProp.maxGridSize[1],
                    deviceProp.maxGridSize[2]);

            // does this device have enough capcacity for the calculation?
            printf("\n*************\n");

            // check memory
            if((unsigned long long) deviceProp.totalGlobalMem < gpu_mem_needed) printf(" FAILURE: Not eneough memeory on device for this calculation! \n");
            else
            {
                printf("Hurrah! This device has enough memory to perform this calculation\n");

                // check # threads

                int threadsPerBlock = deviceProp.maxThreadsPerBlock; // maximal efficiency exists if we use max # threads per block.
                int blocksPerGrid = int(ceil(ncalc / threadsPerBlock)); // need nx*nx threads total
                if(deviceProp.maxThreadsDim[0] >blocksPerGrid) printf("FAILURE: Not enough threads on the device to do this calculation!\n");
                else
                {
                    printf("Hurrah! This device supports enough threads to do this calculation\n");
                    // how many kernels can we run at once on this machine?
                    int n_mem = floor(deviceProp.totalGlobalMem / float(gpu_mem_needed));
                    int n_threads = floor(threadsPerBlock * deviceProp.maxThreadsDim[0]*deviceProp.maxThreadsDim[1] / float(ncalc) ); // max # threads possible?

                    printf("%d %d  \n",  n_threads, deviceProp.maxThreadsDim[0]);

                    int max_kernels = 0;
                    n_mem<n_threads ? max_kernels = n_mem : max_kernels = n_threads;

                    printf(" you can run %d kernels at a time on this device without overloading the resources \n", max_kernels);
                }
            }

        }

        printf("\n------ End CUDA device diagnostics ------\n\n");
    }
int main()
{
	float alpha1= 4646.98;
	float b1= 3749.51;
	float a2=4644.35; 
	float b2=3749.52;
	
	float theta1= acos(sin(b1)*sin(b2) + cos(b1)*cos(b2) *cos(alpha1-a2));
	
	printf("%f\n", b1);
	printf("%f\n", theta1);
	getDeviceDiagnostics(20000, 2);
 	read_the_files(); 
	/*char *ad;
	int *bd;
	const char csize = N*sizeof(char);
	const char isize = N*sizeof(char);
 
	printf("%s", a);
 
	hipMalloc( (void**)&ad, csize ); 
	hipMalloc( (void**)&bd, isize ); 
	hipMemcpy( ad, a, csize, hipMemcpyHostToDevice ); 
	hipMemcpy( bd, b, isize, hipMemcpyHostToDevice ); 
	
	dim3 dimBlock( blocksize, 1 );
	dim3 dimGrid( 1, 1 );
	hello<<<dimGrid, dimBlock>>>(ad, bd);
	hipMemcpy( a, ad, csize, hipMemcpyDeviceToHost ); 
	hipFree( ad );
	hipFree( bd );
	
	printf("%s\n", a);*/
	return EXIT_SUCCESS;
}


