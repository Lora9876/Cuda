
#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<unistd.h>
#include <time.h>
#include<hip/hip_runtime.h>

__global__ void VecAdd(float* A, float* B, float* C, int N)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j= blockDim.y*blockIdx.y + threadIdx.y;
	
	if(j<N)
	C[j] = A[i] + B[i]+5;
}
// CPU Host code
int main(int argc, char *argv[])
{
int N =20;
size_t arraybytes = N * sizeof(float);
// Allocate input vectors h_A and h_B in host memory
float* h_A = (float*)malloc(arraybytes);
float* h_B = (float*)malloc(arraybytes);
float* h_C = (float*)malloc(arraybytes); 
	for(int i=0; i<20; i++)
	{ h_A[i]=i; h_B[i]=i+1;  }
float* d_A; hipMalloc(&d_A, arraybytes);
float* d_B; hipMalloc(&d_B, arraybytes);
float* d_C; hipMalloc(&d_C, arraybytes);
// Copy arrays from host memory to device memory
hipMemcpy(d_A, h_A, arraybytes, hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B, arraybytes, hipMemcpyHostToDevice);
// Invoke kernel
dim3 thr,blocksInGrid;	
 thr.x = 256;
	thr.y=256; 
 blocksInGrid.x = 1;
VecAdd<<<blocksInGrid, thr>>>(d_A, d_B, d_C, N);
// Copy result from device memory to host memory
// h_C contains the result in host memory
hipMemcpy(h_C, d_C, arraybytes, hipMemcpyDeviceToHost);
	
	for(int i=0; i<20; i++)
	{printf("%f  ", h_A[i]); printf("%f  ", h_B[i]);
		printf("%f\n", h_C[i]); }
// Free device memory
hipFree(d_A); hipFree(d_B); hipFree(d_C);
	hipFree(h_A); hipFree(h_B); hipFree(h_C);
// Free host memory ...
}










/*
__global__ void angles(volatile float *a0, volatile float *b1, volatile float *histi)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  //int idy  = threadIdx.y + blockDim.y * blockIdx.y;
  float m;
  __shared__ unsigned int shared[20];
    		// za prvu petlju ocistis uvek
    			
	
   	

  if(idx<20)
	 
	 histi[idx]=  5;
             

	
}
             
             
  int main()
             {
               
               float *a, *b,*a0, *b0,*tmp, *tmp1;
               a= (float*) malloc(20* sizeof(float));
               b= (float*) malloc(20*sizeof(float));
                 tmp= (float*) malloc(20*sizeof(float));
	  a0= (float*) malloc(20* sizeof(float));
               b0= (float*) malloc(20*sizeof(float));
                 tmp1= (float*) malloc(20*sizeof(float));
                
               for(int i=0; i<20;i++)
               { a[i]= i+1; b[i]=i+2; tmp[i]=0;}
            
               cudaMemcpy(a0, a, 20* sizeof(float), cudaMemcpyHostToDevice );
               cudaMemcpy(b0, b,20* sizeof(float), cudaMemcpyHostToDevice );
                dim3 grid, block;
    
               grid.x = 1024; 
                	

                          block.x = 1; 
                angles<<<block, grid>>>(a0, b0, tmp1);
           	   cudaMemcpy(tmp, tmp1, 20*sizeof(float), cudaMemcpyDeviceToHost);
               
               for(int i=0; i<20;i++)
                 printf("%d ", tmp[i]); 
           //    free(a0);
	 // free(a); free(b); free(b0); free(tmp); free(tmp1); 
             return EXIT_SUCCESS;
             
    }
  */
