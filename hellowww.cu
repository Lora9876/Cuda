
#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<unistd.h>
#include <time.h>
#include<hip/hip_runtime.h>

__global__ void angles(volatile float *a0, volatile float *b1, volatile float *histi)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int idy  = threadIdx.y + blockDim.y * blockIdx.y;
  float m;
  __shared__ unsigned int shared[20];
    		// za prvu petlju ocistis uvek
    			if((threadIdx.x==0) && (threadIdx.y==0))
   			 {
       			 for (int i=0;i<20;i++)
         		   shared[i] = 0;
   			 }
	
   	 __syncthreads();

  if(idx<20 && idy<20)
	 m=  a0[idx] + b1[idy];
             
    __syncthreads();
	shared[idx]+=m; 
}
             
             
  int main()
             {
               
               float *a, *b,*a0, *b0,*tmp, *tmp1;
               a= (float*) malloc(20* sizeof(float));
               b= (float*) malloc(20*sizeof(float));
                 tmp= (float*) malloc(20*sizeof(float));
                
               for(int i=0; i<20;i++)
               { a[i]= i+1; b[i]=i+2; tmp[i]=0;}
               hipMalloc((void **) &a0, 20* sizeof(float));
               hipMalloc((void **) &b0, 20* sizeof(float));
                hipMalloc((void **) &tmp1, 20* sizeof(float));
               hipMemset(a0,0,20* sizeof(float));
                hipMemset(b0,0,20* sizeof(float));
               hipMemset(tmp1,0,20* sizeof(float));
               hipMemcpy(a0, a, 20* sizeof(float), hipMemcpyHostToDevice );
               hipMemcpy(b0, b,20* sizeof(float), hipMemcpyHostToDevice );
                dim3 grid, block;
    
               grid.x = 1024; 
                	 grid.y=1024;

                          block.x = 1; 
                angles<<<grid,block>>>(a0, b0, tmp1);
           	   hipMemcpy(tmp, tmp1, 20*sizeof(float), hipMemcpyDeviceToHost);
               
               for(int i=0; i<20;i++)
                 printf("%d ", tmp[i]); 
               
             return EXIT_SUCCESS;
             
    }
  
