
#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<unistd.h>
#include <time.h>
#include "hip/device_functions.h"
#include "hip/hip_runtime.h"

#define SUBMATRIX_SIZE 16384

void getDeviceDiagnostics(int tot_gals, int n_coords){

 ////////////////////////////////////////////////////////////////////////////
    // Now get the info from the device.
    ////////////////////////////////////////////////////////////////////////////
   
        printf("\n------ CUDA device diagnostics ------\n\n");

        
        int nx = SUBMATRIX_SIZE;
        int ncalc = nx * nx;
        int gpu_mem_needed = int(tot_gals * sizeof(float)) * n_coords; // need to allocate ra, dec.
        printf("Requirements: %d calculations and %d bytes memory on the GPU \n\n", ncalc, gpu_mem_needed);

        int deviceCount = 0;
        hipError_t error_id = hipGetDeviceCount(&deviceCount);
        if (error_id != hipSuccess) {
            printf( "hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id) );
        }
        // This function call returns 0 if there are no CUDA capable devices.
        if (deviceCount == 0)
            printf("There is no device supporting CUDA\n");
        else
            printf("Found %d CUDA Capable device(s)\n", deviceCount);


        int dev=0;
        for (dev = 0; dev < deviceCount; ++dev) {
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, dev);
            printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

            printf("  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
                    (float)deviceProp.totalGlobalMem/1048576.0f, (unsigned long long) deviceProp.totalGlobalMem);


            printf("  Warp size:                                     %d\n", deviceProp.warpSize);
            printf("  Maximum number of threads per block:           %d\n", deviceProp.maxThreadsPerBlock);
            printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
                    deviceProp.maxThreadsDim[0],
                    deviceProp.maxThreadsDim[1],
                    deviceProp.maxThreadsDim[2]);
            printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
                    deviceProp.maxGridSize[0],
                    deviceProp.maxGridSize[1],
                    deviceProp.maxGridSize[2]);

            // does this device have enough capcacity for the calculation?
            printf("\n*************\n");

            // check memory
            if((unsigned long long) deviceProp.totalGlobalMem < gpu_mem_needed) printf(" FAILURE: Not eneough memeory on device for this calculation! \n");
            else
            {
                printf("Hurrah! This device has enough memory to perform this calculation\n");

                // check # threads

                int threadsPerBlock = deviceProp.maxThreadsPerBlock; // maximal efficiency exists if we use max # threads per block.
                int blocksPerGrid = int(ceil(ncalc / threadsPerBlock)); // need nx*nx threads total
                if(deviceProp.maxThreadsDim[0] >blocksPerGrid) printf("FAILURE: Not enough threads on the device to do this calculation!\n");
                else
                {
                    printf("Hurrah! This device supports enough threads to do this calculation\n");
                    // how many kernels can we run at once on this machine?
                    int n_mem = floor(deviceProp.totalGlobalMem / float(gpu_mem_needed));
                    int n_threads = floor(threadsPerBlock * deviceProp.maxThreadsDim[0]*deviceProp.maxThreadsDim[1] / float(ncalc) ); // max # threads possible?

                    printf("%d %d  \n",  n_threads, deviceProp.maxThreadsDim[0]);

                    int max_kernels = 0;
                    n_mem<n_threads ? max_kernels = n_mem : max_kernels = n_threads;

                    printf(" you can run %d kernels at a time on this device without overloading the resources \n", max_kernels);
                }
            }

        }

        printf("\n------ End CUDA device diagnostics ------\n\n");
    }

__global__ void VecAdd(float* A, float* B, float* C, int N)
{		float m;
 		int n; 
 		float *addr; 
		int idx = blockDim.x * blockIdx.x + threadIdx.x;
			__shared__ float sab[720]; 
			
 			if(threadIdx.x==0)
			{	
					for(int i=0; i<720; i++) sab[i]=0; 
			}	
	
    __syncthreads();
	
			if (idx<10000)
				for(int i=0; i<10000; i++)
				{
					m= A[idx]*B[i];
					n= int(m);
					sab[n]=sab[n]+1; 
				}
 							
	 __syncthreads();
 		if(threadIdx.x==0)
   	 {
        for(int i=0;i<720;i++)
            C[i+(blockIdx.x*720)]=sab[i];
    }
	
	
	

}
// CPU Host code
int main(int argc, char *argv[])
{
	
 getDeviceDiagnostics(20000,2); 
    

/*int N =10000;
size_t arraybytes = N * sizeof(float);
	size_t arraybytes1 = 720*16384 *sizeof(float);
	size_t l=720*sizeof(float);
// Allocate input vectors h_A and h_B in host memory
float* h_A = (float*)malloc(arraybytes);
float* h_B = (float*)malloc(arraybytes);
float* h_C = (float*)malloc(arraybytes1); 
	float* result=(float*)malloc(l); 
	
	for(int i=0; i<10000; i++)
	{ h_A[i]=1; h_B[i]=1;  }
	h_A[0]=5; h_B[1] =3; 
float* d_A; cudaMalloc(&d_A, arraybytes);
float* d_B; cudaMalloc(&d_B, arraybytes);
float* d_C; cudaMalloc(&d_C, arraybytes1);
// Copy arrays from host memory to device memory
cudaMemcpy(d_A, h_A, arraybytes, cudaMemcpyHostToDevice);
cudaMemcpy(d_B, h_B, arraybytes, cudaMemcpyHostToDevice);
// Invoke kernel
/*dim3 thr,blocksInGrid;	
// thr.x = 256;
	thr.y=256; 
 blocksInGrid.x = 1;
	//dim3 thr(1024), blocksInGrid(100);
	int thr=512;
	int blocksInGrid=32; 
	
VecAdd<<<blocksInGrid, thr>>>(d_A, d_B, d_C, N);
// Copy result from device memory to host memory
// h_C contains the result in host memory
cudaMemcpy(h_C, d_C, arraybytes, cudaMemcpyDeviceToHost);
	
	for(int i=0; i<720*8192; i++)
	{	result[i%720]+= d_C[i]; } 
		
		for(int i=0; i<720*8192; i++)
		printf("%f ", result[i]);   
// Free device memory
cudaFree(d_A); cudaFree(d_B); cudaFree(d_C);
	cudaFree(h_A); cudaFree(h_B); cudaFree(h_C);*/
// Free host memory ...
	
}










/*
__global__ void angles(volatile float *a0, volatile float *b1, volatile float *histi)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  //int idy  = threadIdx.y + blockDim.y * blockIdx.y;
  float m;
  __shared__ unsigned int shared[20];
    		// za prvu petlju ocistis uvek
    			
	
   	

  if(idx<20)
	 
	 histi[idx]=  5;
             

	
}
             
             
  int main()
             {
               
               float *a, *b,*a0, *b0,*tmp, *tmp1;
               a= (float*) malloc(20* sizeof(float));
               b= (float*) malloc(20*sizeof(float));
                 tmp= (float*) malloc(20*sizeof(float));
	  a0= (float*) malloc(20* sizeof(float));
               b0= (float*) malloc(20*sizeof(float));
                 tmp1= (float*) malloc(20*sizeof(float));
                
               for(int i=0; i<20;i++)
               { a[i]= i+1; b[i]=i+2; tmp[i]=0;}
            
               cudaMemcpy(a0, a, 20* sizeof(float), cudaMemcpyHostToDevice );
               cudaMemcpy(b0, b,20* sizeof(float), cudaMemcpyHostToDevice );
                dim3 grid, block;
    
               grid.x = 1024; 
                	

                          block.x = 1; 
                angles<<<block, grid>>>(a0, b0, tmp1);
           	   cudaMemcpy(tmp, tmp1, 20*sizeof(float), cudaMemcpyDeviceToHost);
               
               for(int i=0; i<20;i++)
                 printf("%d ", tmp[i]); 
           //    free(a0);
	 // free(a); free(b); free(b0); free(tmp); free(tmp1); 
             return EXIT_SUCCESS;
             
    }
  */
