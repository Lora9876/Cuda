
#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<unistd.h>
#include <time.h>
#include "hip/device_functions.h"
#include "hip/hip_runtime.h"

#define SUBMATRIX_SIZE 16384
#include <time.h>


            

__global__ void VecAdd(float* A, float* B, int* C,int N,int sum)
{		float m;
 		int n; 
 		
		int idx = blockDim.x * blockIdx.x + threadIdx.x;
		int idy = blockIdx.y*blockDim.y+threadIdx.y;	
 		
 
 			if(idx<sum && idy<sum)
			
			{	for(int i=0;i<sum; i++)
			{		m=A[sum*idx+i]*B[i*sum+idy];
					n=int(m); 
			 		C[idx*10000+ idy*sum+i]=n;
					//mn[n]++;}}
			}}
 			
 						
 //__syncthreads();
 
 

}
// CPU Host code
int main(int argc, char *argv[])
{
	
 

int N =10000;
	int angle; 
size_t arraybytes = N * sizeof(float);
	size_t arraybytes1 = 720*16384 *sizeof(int);
	size_t l=720*sizeof(int);
// Allocate input vectors h_A and h_B in host memory
float* h_A = (float*)malloc(arraybytes);
float* h_B = (float*)malloc(arraybytes);
int* h_C = (int*)malloc(N*arraybytes); 
	
	int* result=(int*)malloc(l); 
	
	for(int i=0; i<10000; i++)
	{ h_A[i]=1; h_B[i]=1;  }
	h_A[0]=5; h_B[1] =3; 
float* d_A; hipMalloc(&d_A, arraybytes);
float* d_B; hipMalloc(&d_B, arraybytes);
int* d_C; hipMalloc(&d_C, N*arraybytes);
	
// Copy arrays from host memory to device memory
hipMemcpy(d_A, h_A, arraybytes, hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B, arraybytes, hipMemcpyHostToDevice);
// Invoke kernel
int NN=50; 
	
	clock_t start, end;
	 dim3 threadsPerBlock(NN, NN);
    dim3 blocksPerGrid(1, 1);
        if (NN*NN > 512){
            threadsPerBlock.x = 512;
            threadsPerBlock.y = 512;
            blocksPerGrid.x = ceil(double(NN)/double(threadsPerBlock.x));
            blocksPerGrid.y = ceil(double(NN)/double(threadsPerBlock.y));
        }
     double cpu_time_used;
     
     start = clock();
    hipMemset(d_C,0,100*arraybytes);
	
 	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N,NN);

hipMemcpy(h_C, d_C, arraybytes, hipMemcpyDeviceToHost);
	
	result[0] = h_C[0] ; 
	result[1]= h_C[1]; 
	result[2]=h_C[3]; 
/*	for(int i=0; i<N*N; i++)
	{	result[0]= h_C[i]; //angle= h_C[i]; result[angle]++; } */

		
	
	
	end = clock();
     cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
	printf("%f\n", cpu_time_used); 
		for(int i=0; i<3; i++)
			//if(result[i]>0)
		printf("%d ", result[i]);   
// Free device memory
hipFree(d_A); hipFree(d_B); hipFree(d_C);
	hipFree(h_A); hipFree(h_B); hipFree(h_C);
// Free host memory ...
	
}








