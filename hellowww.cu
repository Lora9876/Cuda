
#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<unistd.h>
#include <time.h>
#include "hip/device_functions.h"
#include "hip/hip_runtime.h"


__global__ void VecAdd(float* A, float* B, float* C, int N)
{		float m;
 		int n; 
 		float *addr; 
		int idx = blockDim.x * blockIdx.x + threadIdx.x;
			__shared__ float sab[720]; 
			
 			if(threadIdx.x==0)
			{	
					for(int i=0; i<720; i++) sab[i]=0; 
			}	
	
    __syncthreads();
	
			if (idx<10000)
				for(int i=0; i<10000; i++)
				{
					m= A[idx]*B[i];
					n= int(m); addr=&sab[n];  
 					atomicAdd(addr,1.0) ;
				}
 							
	 __syncthreads();
 		if(threadIdx.x==0)
   	 {
        for(int i=0;i<720;i++)
            C[i+(blockIdx.x*720)]=sab[i];
    }
	
	
	

}
// CPU Host code
int main(int argc, char *argv[])
{
	
	 
    

int N =10000;
size_t arraybytes = N * sizeof(float);
	size_t arraybytes1 = 720*16384 *sizeof(float);
	size_t l=720*sizeof(float);
// Allocate input vectors h_A and h_B in host memory
float* h_A = (float*)malloc(arraybytes);
float* h_B = (float*)malloc(arraybytes);
float* h_C = (float*)malloc(arraybytes1); 
	float* result=(float*)malloc(l); 
	
	for(int i=0; i<10000; i++)
	{ h_A[i]=1; h_B[i]=1;  }
	h_A[0]=5; h_B[1] =3; 
float* d_A; hipMalloc(&d_A, arraybytes);
float* d_B; hipMalloc(&d_B, arraybytes);
float* d_C; hipMalloc(&d_C, arraybytes1);
// Copy arrays from host memory to device memory
hipMemcpy(d_A, h_A, arraybytes, hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B, arraybytes, hipMemcpyHostToDevice);
// Invoke kernel
/*dim3 thr,blocksInGrid;	
// thr.x = 256;
	thr.y=256; 
 blocksInGrid.x = 1;*/
	//dim3 thr(1024), blocksInGrid(100);
	int thr=512;
	int blocksInGrid=32; 
	
VecAdd<<<blocksInGrid, thr>>>(d_A, d_B, d_C, N);
// Copy result from device memory to host memory
// h_C contains the result in host memory
hipMemcpy(h_C, d_C, arraybytes, hipMemcpyDeviceToHost);
	
	for(int i=0; i<720*16384; i++)
	{	result[i%720]+= d_C[i]; } 
		
		for(int i=0; i<720*16384; i++)
		printf("%f ", result[i]);   
// Free device memory
hipFree(d_A); hipFree(d_B); hipFree(d_C);
	hipFree(h_A); hipFree(h_B); hipFree(h_C);
// Free host memory ...
}










/*
__global__ void angles(volatile float *a0, volatile float *b1, volatile float *histi)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  //int idy  = threadIdx.y + blockDim.y * blockIdx.y;
  float m;
  __shared__ unsigned int shared[20];
    		// za prvu petlju ocistis uvek
    			
	
   	

  if(idx<20)
	 
	 histi[idx]=  5;
             

	
}
             
             
  int main()
             {
               
               float *a, *b,*a0, *b0,*tmp, *tmp1;
               a= (float*) malloc(20* sizeof(float));
               b= (float*) malloc(20*sizeof(float));
                 tmp= (float*) malloc(20*sizeof(float));
	  a0= (float*) malloc(20* sizeof(float));
               b0= (float*) malloc(20*sizeof(float));
                 tmp1= (float*) malloc(20*sizeof(float));
                
               for(int i=0; i<20;i++)
               { a[i]= i+1; b[i]=i+2; tmp[i]=0;}
            
               cudaMemcpy(a0, a, 20* sizeof(float), cudaMemcpyHostToDevice );
               cudaMemcpy(b0, b,20* sizeof(float), cudaMemcpyHostToDevice );
                dim3 grid, block;
    
               grid.x = 1024; 
                	

                          block.x = 1; 
                angles<<<block, grid>>>(a0, b0, tmp1);
           	   cudaMemcpy(tmp, tmp1, 20*sizeof(float), cudaMemcpyDeviceToHost);
               
               for(int i=0; i<20;i++)
                 printf("%d ", tmp[i]); 
           //    free(a0);
	 // free(a); free(b); free(b0); free(tmp); free(tmp1); 
             return EXIT_SUCCESS;
             
    }
  */
