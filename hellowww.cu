
#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<unistd.h>
#include <time.h>
#include<hip/hip_runtime.h>

__global__ void VecAdd(float* A, float* B, float* C, int N)
{
int i = blockDim.x * blockIdx.x + threadIdx.x;
	int j= threadIdx.y; 
	if (i<50)
		C[i]= 250; C[j]= C[j]+ 126; 

}
// CPU Host code
int main(int argc, char *argv[])
{
	
	 int numBlocks;        // Occupancy in terms of active blocks
    int blockSize = 32;

    // These variables are used to convert occupancy to warps
    int device;
    hipDeviceProp_t prop;
    int activeWarps;
    int maxWarps;

    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    
    

   // activeWarps = numBlocks * blockSize / prop.warpSize;
   // maxWarps = prop.maxThreadsPerMultiProcessor / prop.warpSize;

   printf("%d\n", prop.warpSize); 
	printf("%d\n", prop.maxThreadsPerMultiProcessor); 
    

int N =50;
size_t arraybytes = N * sizeof(float);
// Allocate input vectors h_A and h_B in host memory
float* h_A = (float*)malloc(arraybytes);
float* h_B = (float*)malloc(arraybytes);
float* h_C = (float*)malloc(arraybytes); 
	for(int i=0; i<50; i++)
	{ h_A[i]=i; h_B[i]=i+1;  }
float* d_A; hipMalloc(&d_A, arraybytes);
float* d_B; hipMalloc(&d_B, arraybytes);
float* d_C; hipMalloc(&d_C, arraybytes);
// Copy arrays from host memory to device memory
hipMemcpy(d_A, h_A, arraybytes, hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B, arraybytes, hipMemcpyHostToDevice);
// Invoke kernel
/*dim3 thr,blocksInGrid;	
// thr.x = 256;
	thr.y=256; 
 blocksInGrid.x = 1;*/
	dim3 thr(32,32), blocksInGrid(1);
	
VecAdd<<<blocksInGrid, thr>>>(d_A, d_B, d_C, N);
// Copy result from device memory to host memory
// h_C contains the result in host memory
hipMemcpy(h_C, d_C, arraybytes, hipMemcpyDeviceToHost);
	
	for(int i=0; i<50; i++)
	{printf("%f  ", h_A[i]); printf("%f  ", h_B[i]);
		printf("%f\n", h_C[i]); h_C[i]=0;  }
// Free device memory
hipFree(d_A); hipFree(d_B); hipFree(d_C);
	hipFree(h_A); hipFree(h_B); hipFree(h_C);
// Free host memory ...
}










/*
__global__ void angles(volatile float *a0, volatile float *b1, volatile float *histi)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  //int idy  = threadIdx.y + blockDim.y * blockIdx.y;
  float m;
  __shared__ unsigned int shared[20];
    		// za prvu petlju ocistis uvek
    			
	
   	

  if(idx<20)
	 
	 histi[idx]=  5;
             

	
}
             
             
  int main()
             {
               
               float *a, *b,*a0, *b0,*tmp, *tmp1;
               a= (float*) malloc(20* sizeof(float));
               b= (float*) malloc(20*sizeof(float));
                 tmp= (float*) malloc(20*sizeof(float));
	  a0= (float*) malloc(20* sizeof(float));
               b0= (float*) malloc(20*sizeof(float));
                 tmp1= (float*) malloc(20*sizeof(float));
                
               for(int i=0; i<20;i++)
               { a[i]= i+1; b[i]=i+2; tmp[i]=0;}
            
               cudaMemcpy(a0, a, 20* sizeof(float), cudaMemcpyHostToDevice );
               cudaMemcpy(b0, b,20* sizeof(float), cudaMemcpyHostToDevice );
                dim3 grid, block;
    
               grid.x = 1024; 
                	

                          block.x = 1; 
                angles<<<block, grid>>>(a0, b0, tmp1);
           	   cudaMemcpy(tmp, tmp1, 20*sizeof(float), cudaMemcpyDeviceToHost);
               
               for(int i=0; i<20;i++)
                 printf("%d ", tmp[i]); 
           //    free(a0);
	 // free(a); free(b); free(b0); free(tmp); free(tmp1); 
             return EXIT_SUCCESS;
             
    }
  */
