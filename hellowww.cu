#include "hip/hip_runtime.h"

#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<unistd.h>
#include <time.h>
#include "hip/device_functions.h"
#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#define SUBMATRIX_SIZE 16384
#include <time.h>


            

__global__ void VecAdd(float* A, float* B, int* C, int* D,int N,int sum)
{		float m;
 		int n; 
 		
		int idx = blockDim.x * blockIdx.x + threadIdx.x;
		int idy = blockIdx.y*blockDim.y+threadIdx.y;	
		__shared__ int mn[720];
 			if(threadIdx.x==0)
				for(int i=0; i<720; i++)
					mn[i]=0; 
    __syncthreads();
	
 
 			if(idx<sum && idy<sum)
			
			{	for(int i=0;i<sum; i++)
			{		m=A[sum*idx+i]*B[i*sum+idy];
					n=int(m); 
			 	D[sum*idx+i]=n;
					//mn[n]++;}}
			}}
 			
 							
	__syncthreads();

    if(threadIdx.x==0 && threadIdx.y=0)
    {
        for(int i=0;i<10000;i++)
	{
          n=D[i];
		C[n]++; 
	}
    }
 

}
// CPU Host code
int main(int argc, char *argv[])
{
	
 

int N =10000;
size_t arraybytes = N * sizeof(float);
	size_t arraybytes1 = 720*16384 *sizeof(int);
	size_t l=100*sizeof(int);
// Allocate input vectors h_A and h_B in host memory
float* h_A = (float*)malloc(arraybytes);
float* h_B = (float*)malloc(arraybytes);
int* h_C = (int*)malloc(arraybytes); 
	int* h_D = (int*)malloc(arraybytes); 
	int* result=(int*)malloc(l); 
	
	for(int i=0; i<10000; i++)
	{ h_A[i]=1; h_B[i]=1;  }
	h_A[0]=5; h_B[1] =3; 
float* d_A; hipMalloc(&d_A, arraybytes);
float* d_B; hipMalloc(&d_B, arraybytes);
int* d_C; hipMalloc(&d_C, arraybytes);
	int* d_D; hipMalloc(&d_D, arraybytes);
// Copy arrays from host memory to device memory
hipMemcpy(d_A, h_A, arraybytes, hipMemcpyHostToDevice);
hipMemcpy(d_B, h_B, arraybytes, hipMemcpyHostToDevice);
// Invoke kernel
int NN=50; 
	
	clock_t start, end;
	 dim3 threadsPerBlock(NN, NN);
    dim3 blocksPerGrid(1, 1);
        if (NN*NN > 512){
            threadsPerBlock.x = 512;
            threadsPerBlock.y = 512;
            blocksPerGrid.x = ceil(double(NN)/double(threadsPerBlock.x));
            blocksPerGrid.y = ceil(double(NN)/double(threadsPerBlock.y));
        }
     double cpu_time_used;
     
     start = clock();
    hipMemset(d_C,0,arraybytes);
	 hipMemset(d_D,0,arraybytes);
 	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C,d_D, N,NN);

hipMemcpy(h_C, d_C, arraybytes, hipMemcpyDeviceToHost);
	
	for(int i=0; i<10000; i++)
	{	result[i%100]+= h_C[i]; } 

		
	
	
	end = clock();
     cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
	printf("%f\n", cpu_time_used); 
		for(int i=0; i<720; i++)
			//if(result[i]>0)
		printf("%d ", result[i]);   
// Free device memory
hipFree(d_A); hipFree(d_B); hipFree(d_C);
	hipFree(h_A); hipFree(h_B); hipFree(h_C);
// Free host memory ...
	
}










/*
__global__ void angles(volatile float *a0, volatile float *b1, volatile float *histi)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  //int idy  = threadIdx.y + blockDim.y * blockIdx.y;
  float m;
  __shared__ unsigned int shared[20];
    		// za prvu petlju ocistis uvek
    			
	
   	

  if(idx<20)
	 
	 histi[idx]=  5;
             

	
}
             
             
  int main()
             {
               
               float *a, *b,*a0, *b0,*tmp, *tmp1;
               a= (float*) malloc(20* sizeof(float));
               b= (float*) malloc(20*sizeof(float));
                 tmp= (float*) malloc(20*sizeof(float));
	  a0= (float*) malloc(20* sizeof(float));
               b0= (float*) malloc(20*sizeof(float));
                 tmp1= (float*) malloc(20*sizeof(float));
                
               for(int i=0; i<20;i++)
               { a[i]= i+1; b[i]=i+2; tmp[i]=0;}
            
               hipMemcpy(a0, a, 20* sizeof(float), hipMemcpyHostToDevice );
               hipMemcpy(b0, b,20* sizeof(float), hipMemcpyHostToDevice );
                dim3 grid, block;
    
               grid.x = 1024; 
                	

                          block.x = 1; 
                angles<<<block, grid>>>(a0, b0, tmp1);
           	   hipMemcpy(tmp, tmp1, 20*sizeof(float), hipMemcpyDeviceToHost);
               
               for(int i=0; i<20;i++)
                 printf("%d ", tmp[i]); 
           //    free(a0);
	 // free(a); free(b); free(b0); free(tmp); free(tmp1); 
             return EXIT_SUCCESS;
             
    }
  */
