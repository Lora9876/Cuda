
#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<unistd.h>
#include <time.h>
#include<hip/hip_runtime.h>

__global__ void angles(volatile float *a0, volatile float *b1, volatile float *histi)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  //int idy  = threadIdx.y + blockDim.y * blockIdx.y;
  float m;
  __shared__ unsigned int shared[20];
    		// za prvu petlju ocistis uvek
    			
	
   	

  if(idx<20)
	  for(int i=0; i<20;i++) 
	 m=  a0[idx] + b1[i];
             
    __syncthreads();
	if(idx<20)
	shared[idx]+=m; 
	
}
             
             
  int main()
             {
               
               float *a, *b,*a0, *b0,*tmp, *tmp1;
               a= (float*) malloc(20* sizeof(float));
               b= (float*) malloc(20*sizeof(float));
                 tmp= (float*) malloc(20*sizeof(float));
                
               for(int i=0; i<20;i++)
               { a[i]= i+1; b[i]=i+2; tmp[i]=0;}
            
               hipMemcpy(a0, a, 20* sizeof(float), hipMemcpyHostToDevice );
               hipMemcpy(b0, b,20* sizeof(float), hipMemcpyHostToDevice );
                dim3 grid, block;
    
               grid.x = 1024; 
                	

                          block.x = 1; 
                angles<<<block, grid>>>(a0, b0, tmp1);
           	   hipMemcpy(tmp, tmp1, 20*sizeof(float), hipMemcpyDeviceToHost);
               
               for(int i=0; i<20;i++)
                 printf("%d ", tmp[i]); 
           //    free(a0);
	 // free(a); free(b); free(b0); free(tmp); free(tmp1); 
             return EXIT_SUCCESS;
             
    }
  
