
#include<stdio.h>
#include<string.h>
#include<stdlib.h>
#include<math.h>
#include<unistd.h>
#include <time.h>
#include "hip/device_functions.h"
#include "hip/hip_runtime.h"

#define SUBMATRIX_SIZE 16384
#include <time.h>

#define PI 3.14159265
            
/*
__global__ void VecAdd(volatile float* A, volatile float* B,volatile int* C,int N,int sum)
{		float m;
 		int n; 
 		
		int idx = blockDim.x * blockIdx.x + threadIdx.x;
		int idy = blockIdx.y*blockDim.y+threadIdx.y;	
 		
 
 			if(idx<sum && idy<sum)
			
			{	for(int i=0;i<sum; i++)
					for(int j=0; j<sum; j++) 
			{		m=A[sum*idx+i]*B[i*sum+j];
					n=int(m); 
					// C[1]++;
			 		C[idx*10000+ idy*sum+i]++;
					//mn[n]++;}}
			}}
 			
 						
// __syncthreads();
 
 */


// CPU Host code
int main(int argc, char *argv[])
{
	float fix =0.00029074074; 
	float a0=2700; float a1= 2700; float b0=1800; float b1=3600; 
	float m=acosf((sin(b0*fix)*sin(b1*fix))+ cos(b0*fix)*cos(b1*fix)*cos((a1-a0)*fix));
	
	m=m*180/PI; 
	m=m/0.25;
	int n;
	n= (int) m;
	
	printf("%d", n);
	
	//angle = acosf((sin(b0[idx]) *sin(b1[i]) + cos(b0[idx]) * cos(b1[i]) * cos(a0[idx]*-a1[i]))*1/val);
	
 /*

int N =10000;
	int angle; 
size_t arraybytes = N * sizeof(float);
	size_t arraybytes1 = N*N *sizeof(int);
	size_t l=720*sizeof(int);
// Allocate input vectors h_A and h_B in host memory
float* h_A = (float*)malloc(arraybytes);
float* h_B = (float*)malloc(arraybytes);
int* h_C = (int*)malloc(arraybytes1); 
	
	int* result=(int*)malloc(l); 
	
	for(int i=0; i<10000; i++)
	{ h_A[i]=1; h_B[i]=1;  }
	h_A[0]=5; h_B[1] =3; 
float* d_A; cudaMalloc(&d_A, arraybytes);
float* d_B; cudaMalloc(&d_B, arraybytes);
int* d_C; cudaMalloc(&d_C, arraybytes1);
	
// Copy arrays from host memory to device memory
cudaMemcpy(d_A, h_A, arraybytes, cudaMemcpyHostToDevice);
cudaMemcpy(d_B, h_B, arraybytes, cudaMemcpyHostToDevice);
// Invoke kernel
int NN=100; 
	
	clock_t start, end;
	 dim3 threadsPerBlock(128, 128);
    dim3 blocksPerGrid(1, 1);
        /*if (NN*NN > 512){
            threadsPerBlock.x = 512;
            threadsPerBlock.y = 512;
            blocksPerGrid.x = ceil(double(NN)/double(threadsPerBlock.x));
            blocksPerGrid.y = ceil(double(NN)/double(threadsPerBlock.y));
        }
     double cpu_time_used;
     
     start = clock();
    cudaMemset(d_C,0,arraybytes1);
	
 	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N,NN);

cudaMemcpy(h_C, d_C, arraybytes, cudaMemcpyDeviceToHost);
	
	result[0] = h_C[0] ; 
	result[1]= h_C[1]; 
	result[2]=h_C[3]; 
/*	for(int i=0; i<N*N; i++)
	{	result[0]= h_C[i]; //angle= h_C[i]; result[angle]++; } 

		
	
	
	end = clock();
     cpu_time_used = ((double) (end - start)) / CLOCKS_PER_SEC;
	printf("%f\n", cpu_time_used); 
		for(int i=0; i<3; i++)
			//if(result[i]>0)
		printf("%d ", result[i]);   
// Free device memory
cudaFree(d_A); cudaFree(d_B); cudaFree(d_C);
	cudaFree(h_A); cudaFree(h_B); cudaFree(h_C);
// Free host memory ...
	*/
}








